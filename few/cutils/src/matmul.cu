#include "hip/hip_runtime.h"
// Code for matrix operations for roman neural network in Fast EMRI Waveforms

// Copyright (C) 2020 Michael L. Katz, Alvin J.K. Chua, Niels Warburton, Scott A. Hughes
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <https://www.gnu.org/licenses/>.

#include "stdio.h"
#include "matmul.hh"
#include "cuda_complex.hpp"
#include <chrono>
#include "global.h"

using namespace std;

using namespace std::chrono;

// adjust imports for CUDA
#ifdef __HIPCC__
#include "hipblas.h"
#else
#include <gsl/gsl_cblas.h>
#endif

#define NUM_THREADS 256

// activation function
// fixed 0.2 in leaky end
CUDA_CALLABLE_MEMBER double LeakyReLU(double x){
     double out = (x >= 0.0) ? x : 0.2*x;
     return out;
}

// funciton for adding bias and then passing through activation
CUDA_KERNEL
void add_bias_relu(double *C, double *bias, int input_len, int dim2)
{

    // adjust loop boundaries in CUDA
    #ifdef __HIPCC__
    int start1 = blockIdx.x * blockDim.x + threadIdx.x;
    int end1 = input_len;
    int diff1 = blockDim.x * gridDim.x;

    int start2 = blockIdx.y * blockDim.y + threadIdx.y;
    int end2 = dim2;
    int diff2 = blockDim.y * gridDim.y;

    #else

    int start1 = 0;
    int end1 = input_len;
    int diff1 = 1;

    int start2 = 0;
    int end2 = dim2;
    int diff2 = 1;


    #endif
    for (int i = start1;
         i < end1;
         i += diff1)
    {

        for (int j = start2;
          j < end2;
          j += diff2)
        {

            C[input_len*j + i] = LeakyReLU(C[input_len*j + i] + bias[j]);

        }
    }
}

// funciton for adding bias and WITHOUT passing through activation
CUDA_KERNEL
void add_bias(double *C, double *bias, int input_len, int dim2){


    #ifdef __HIPCC__
    int start1 = blockIdx.x * blockDim.x + threadIdx.x;
    int end1 = input_len;
    int diff1 = blockDim.x * gridDim.x;

    int start2 = blockIdx.y * blockDim.y + threadIdx.y;
    int end2 = dim2;
    int diff2 = blockDim.y * gridDim.y;

    #else

    int start1 = 0;
    int end1 = input_len;
    int diff1 = 1;

    int start2 = 0;
    int end2 = dim2;
    int diff2 = 1;


    #endif
    for (int i = start1;
         i < end1;
         i += diff1)
    {

        for (int j = start2;
          j < end2;
          j += diff2)
        {

            C[input_len*j + i] = C[input_len*j + i] + bias[j];
        }
    }
}

// perform matrix calculations in blas for a neural network layer
void neural_layer(double *mat_out, double *mat_in, double *weight, double *bias, int m, int k, int n, int run_relu)
{
    #ifdef __HIPCC__
       hipblasHandle_t handle;

       char * status;
       hipblasStatus_t stat;
       double alpha = 1.0;
       double beta = 0.0;
       stat = hipblasCreate(&handle);
          if (stat != HIPBLAS_STATUS_SUCCESS) {
                  printf ("CUBLAS initialization failed\n");
                  exit(0);
              }

        // matrix multiplication
       stat = hipblasDgemm(handle,
                              HIPBLAS_OP_N, HIPBLAS_OP_N,
                              m, n, k,
                              &alpha,
                              mat_in, m,
                              weight, k,
                              &beta,
                              mat_out, m);

       if (stat != HIPBLAS_STATUS_SUCCESS) {
              printf ("CUBLAS initialization failed\n");
              exit(0);
          }

      stat = hipblasDestroy(handle);
         if (stat != HIPBLAS_STATUS_SUCCESS) {
                 printf ("CUBLAS initialization failed\n");
                 exit(0);
             }

    // Add the bias and activate, except in last layer do not activate
     int num_threads = 256;
     int num_blocks = std::ceil((m + num_threads -1)/num_threads);
     dim3 gridDim(num_blocks, n);

     if (run_relu){
         add_bias_relu<<<gridDim, num_threads>>>(mat_out, bias, m, n);
     } else {
         add_bias<<<gridDim, num_threads>>>(mat_out, bias, m, n);
     }
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());

    #else

    // perform calculations in cblas
     cblas_dgemm (CblasColMajor,
               CblasNoTrans, CblasNoTrans, m, n, k,
                1.0, mat_in, m, weight, k, 0.0, mat_out, m);

    if (run_relu){
        add_bias_relu(mat_out, bias, m, n);
    } else {
        add_bias(mat_out, bias, m, n);
    }

    #endif
}

// take the output of the neural net and conver it from (re_1,..,re_n, im_1, ..., im_n)
// to imaginary
CUDA_KERNEL
void form_complex_output(cmplx *complex_output, double *nn_output, int input_len, int break_index,
                          double transform_factor_inv){

  cmplx temp(0.0, 0.0);

  #ifdef __HIPCC__
  int start1 = blockIdx.x * blockDim.x + threadIdx.x;
  int end1 = input_len;
  int diff1 = blockDim.x * gridDim.x;

  int start2 = blockIdx.y * blockDim.y + threadIdx.y;
  int end2 = break_index;
  int diff2 = blockDim.y * gridDim.y;

  #else

  int start1 = 0;
  int end1 = input_len;
  int diff1 = 1;

  int start2 = 0;
  int end2 = break_index;
  int diff2 = 1;


  #endif
  for (int i = start1;
       i < end1;
       i += diff1){

   for (int ind = start2;
        ind < end2;
        ind += diff2){

            // break index tells how many real entries or imaginary entries
            temp = cmplx(nn_output[ind*input_len + i], nn_output[(break_index+ind)*input_len + i]);
            complex_output[ind*input_len + i] = temp*transform_factor_inv;
         }
  }
}

// post neural net transform from reduced basis back to full amplitude basis
void transform_output(cmplx *teuk_modes, cmplx *transform_matrix, cmplx *nn_output_mat, double *C,
                      int input_len, int break_index, double transform_factor_inv,
                      int num_teuk_modes){

  int m=input_len, k=break_index, n=num_teuk_modes;
  #ifdef __HIPCC__
  int num_blocks = std::ceil((input_len + NUM_THREADS -1)/NUM_THREADS);
  dim3 gridDim(num_blocks, break_index);

  // form the complex array of neural net outputs
  form_complex_output<<<gridDim, NUM_THREADS>>>(nn_output_mat, C, input_len, break_index, transform_factor_inv);
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());


  char * status;
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
  hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);
  stat = hipblasCreate(&handle);
     if (stat != HIPBLAS_STATUS_SUCCESS) {
             printf ("CUBLAS initialization failed\n");
             exit(0);
         }

  // project back onto amplitude basis
  stat = hipblasZgemm(handle,
                         HIPBLAS_OP_N, HIPBLAS_OP_N,
                         m, n, k,
                         &alpha,
                         (hipDoubleComplex*)nn_output_mat, m,
                         (hipDoubleComplex*)transform_matrix, k,
                         &beta,
                         (hipDoubleComplex*)teuk_modes, m);

   status = _cudaGetErrorEnum(stat);
    hipDeviceSynchronize();

    stat = hipblasDestroy(handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
            exit(0);
        }

   #else

   const cmplx alpha(1.0, 0.0);
   const cmplx beta(0.0, 0.0);

    // form the complex array of neural net outputs
   form_complex_output(nn_output_mat, C, input_len, break_index, transform_factor_inv);

   // transform to amplitude basis
   cblas_zgemm (CblasColMajor,
                  CblasNoTrans, CblasNoTrans, m, n, k,
                  (void*)&alpha, (void*)nn_output_mat, m, (void*)transform_matrix, k, (void*)&beta, (void*)teuk_modes, m);
   #endif
}
