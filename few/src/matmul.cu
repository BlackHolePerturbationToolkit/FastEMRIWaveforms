#include "hip/hip_runtime.h"
#include "stdio.h"
#include <gsl/gsl_cblas.h>
#include "matmul.hh"
#include "cuda_complex.hpp"
#include <chrono>

using namespace std::chrono;

#ifdef __HIPCC__
#include "hipblas.h"
#endif

#define NUM_THREADS 256


__device__ __host__ double LeakyReLU(double x){
     double out = (x >= 0.0) ? x : 0.2*x;
     return out;
}

__global__
void add_bias_relu(double *C, double *bias, int input_len, int dim2){

 for (int j = blockIdx.y * blockDim.y + threadIdx.y;
      j < dim2;
      j += blockDim.y * gridDim.y){

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
     i < input_len;
     i += blockDim.x * gridDim.x){

        C[input_len*j + i] = LeakyReLU(C[input_len*j + i] + bias[j]);

  }
}
}

__global__
void add_bias(double *C, double *bias, int input_len, int dim2){

 for (int j = blockIdx.y * blockDim.y + threadIdx.y;
      j < dim2;
      j += blockDim.y * gridDim.y){

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
     i < input_len;
     i += blockDim.x * gridDim.x){

       C[input_len*j + i] = C[input_len*j + i] + bias[j];
  }
}
}


void neural_layer(double *mat_out, double *mat_in, double *weight, double *bias, int m, int k, int n, int run_relu)
{
    //high_resolution_clock::time_point t1 = high_resolution_clock::now();
    #ifdef __HIPCC__
       hipblasHandle_t handle;

       char * status;
       hipblasStatus_t stat;
       double alpha = 1.0;
       double beta = 0.0;
       stat = hipblasCreate(&handle);
          if (stat != HIPBLAS_STATUS_SUCCESS) {
                  printf ("CUBLAS initialization failed\n");
                  exit(0);
              }

       stat = hipblasDgemm(handle,
                              HIPBLAS_OP_N, HIPBLAS_OP_N,
                              m, n, k,
                              &alpha,
                              mat_in, m,
                              weight, k,
                              &beta,
                              mat_out, m);

       if (stat != HIPBLAS_STATUS_SUCCESS) {
              printf ("CUBLAS initialization failed\n");
              exit(0);
          }

      stat = hipblasDestroy(handle);
         if (stat != HIPBLAS_STATUS_SUCCESS) {
                 printf ("CUBLAS initialization failed\n");
                 exit(0);
             }

     int num_threads = 256;
     int num_blocks = std::ceil((m + num_threads -1)/num_threads);
     dim3 gridDim(num_blocks, n);

     if (run_relu){
         add_bias_relu<<<gridDim, num_threads>>>(mat_out, bias, m, n);
     } else {
         add_bias<<<gridDim, num_threads>>>(mat_out, bias, m, n);
     }
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());

    #else
     cblas_dgemm (CblasColMajor,
                    CblasNoTrans, CblasNoTrans, m, n, k,
                    1.0, mat_in, m, weight, k, 0.0, mat_out, m);


    #endif

    //high_resolution_clock::time_point t2 = high_resolution_clock::now();
    //duration<double> time_span = duration_cast<duration<double> >(t2 - t1);
    //printf("# Computing the inspiral took (%d,%d,%d): %lf\n", m,k,n,time_span.count());
}


__global__
void form_complex_output(cmplx *complex_output, double *nn_output, int input_len, int break_index,
                          double transform_factor_inv){

  cmplx temp(0.0, 0.0);
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < input_len;
       i += blockDim.x * gridDim.x){

   for (int ind = blockIdx.y * blockDim.y + threadIdx.y;
        ind < break_index;
        ind += blockDim.y * gridDim.y){
            temp = cmplx(nn_output[ind*input_len + i], nn_output[(break_index+ind)*input_len + i]);
            complex_output[ind*input_len + i] = temp*transform_factor_inv;

            if ((i == 40) && (ind == 0)) printf("%e + %e j\n", complex_output[ind*input_len + i].real(), complex_output[ind*input_len + i].imag());
         }
  }
}

void transform_output(cmplx *teuk_modes, cmplx *transform_matrix, cmplx *nn_output_mat, double *C,
                      int input_len, int break_index, double transform_factor_inv,
                      int num_teuk_modes){
  int num_blocks = std::ceil((input_len + NUM_THREADS -1)/NUM_THREADS);
  dim3 gridDim(num_blocks, break_index);
  form_complex_output<<<gridDim, NUM_THREADS>>>(nn_output_mat, C, input_len, break_index, transform_factor_inv);
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());


  int m=input_len, k=break_index, n=num_teuk_modes;
  char * status;
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
  hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);
  stat = hipblasCreate(&handle);
     if (stat != HIPBLAS_STATUS_SUCCESS) {
             printf ("CUBLAS initialization failed\n");
             exit(0);
         }

  stat = hipblasZgemm(handle,
                         HIPBLAS_OP_N, HIPBLAS_OP_N,
                         m, n, k,
                         &alpha,
                         (hipDoubleComplex*)nn_output_mat, m,
                         (hipDoubleComplex*)transform_matrix, k,
                         &beta,
                         (hipDoubleComplex*)teuk_modes, m);

   status = _cudaGetErrorEnum(stat);
    hipDeviceSynchronize();

    if (stat != HIPBLAS_STATUS_SUCCESS) {
            exit(0);
        }

}
