#include "hip/hip_runtime.h"
/*
This is the central piece of code. This file implements a class
that takes data in on the cpu side, copies
it to the gpu, and exposes functions that let
you perform actions with the GPU

This class will get translated into python via cython
*/

#include <kernel.hh>
#include <manager.hh>
#include <assert.h>
#include <iostream>
#include "global.h"
#include <complex>
#include "hip/hip_complex.h"
#include "elliptic.hh"
#include "ylm.hh"
#include "FluxInspiral.hh"
#include "interpolate.hh"

using namespace std;

#define NUM_THREADS 256

FastEMRIWaveforms::FastEMRIWaveforms (int time_batch_size_, int num_layers_, int *dim1_, int *dim2_,
    fod *flatten_weight_matrix, fod *flattened_bias_matrix,
    cmplx*transform_matrix, int trans_dim1_, int trans_dim2_, fod transform_factor_,
    int break_index_,
    int *l_, int *m_, int *n_,
    int max_input_len_, int num_l_m_, int num_n_, fod delta_t_,
    int max_init_len_, double int_err_, fod tol_)
{
    max_input_len = max_input_len_;
    time_batch_size = time_batch_size_;
    num_layers = num_layers_;
    dim1 = dim1_;
    dim2 = dim2_;
    break_index = break_index_;
    delta_t = delta_t_;

    trans_dim1 = trans_dim1_;
    trans_dim2 = trans_dim2_;
    transform_factor = transform_factor_;

    num_n = num_n_;
    num_l_m = num_l_m_;

    int_err = int_err_;

    l_arr = l_;
    m_arr = m_;
    max_init_len = max_init_len_;

    num_teuk_modes = trans_dim2;

    d_transform_factor_inv = make_hipDoubleComplex(1./transform_factor, 0.0);

    d_layers_matrix = new fod*[num_layers];
    d_layers_bias = new fod*[num_layers];


    int start_int_weights = 0;
    int start_int_bias = 0;
    for (int i=0; i<num_layers; i++){
      gpuErrchk(hipMalloc(&(d_layers_matrix[i]), dim1[i]*dim2[i]*sizeof(fod)));
      gpuErrchk(hipMemcpy(d_layers_matrix[i], &flatten_weight_matrix[start_int_weights], dim1[i]*dim2[i]*sizeof(fod), hipMemcpyHostToDevice));

      gpuErrchk(hipMalloc(&d_layers_bias[i], dim2[i]*sizeof(fod)));
      gpuErrchk(hipMemcpy(d_layers_bias[i], &flattened_bias_matrix[start_int_bias], dim2[i]*sizeof(fod), hipMemcpyHostToDevice));

      start_int_weights += dim1[i]*dim2[i];
      start_int_bias += dim2[i];
    }

    gpuErrchk(hipMalloc(&d_transform_matrix, trans_dim1*trans_dim2*sizeof(hipDoubleComplex)));
    gpuErrchk(hipMemcpy(d_transform_matrix, transform_matrix, trans_dim1*trans_dim2*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    // allocate buffer matrix
    dim_max = 0;
    for (int i=0; i<num_layers; i++){
        if (dim2[i] > dim_max) dim_max = dim2[i];
    }
    if (dim1[0] > dim_max) dim_max = dim1[0];

    gpuErrchk(hipMalloc(&d_l, num_teuk_modes*sizeof(int)));
    gpuErrchk(hipMalloc(&d_m, num_teuk_modes*sizeof(int)));
    gpuErrchk(hipMalloc(&d_n, num_teuk_modes*sizeof(int)));

    gpuErrchk(hipMemcpy(d_l, l_, num_teuk_modes*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_m, m_, num_teuk_modes*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_n, n_, num_teuk_modes*sizeof(int), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_Ylms, 2*num_l_m*sizeof(hipDoubleComplex)));
    Ylms = new cmplx[2*num_l_m];

    gpuErrchk(hipMalloc(&d_C, max_init_len*dim_max*sizeof(fod)));

    int complex_dim = (int)((fod) dim2[num_layers - 1]/ 2.0);
    gpuErrchk(hipMalloc(&d_nn_output_mat, complex_dim*max_init_len*sizeof(hipDoubleComplex)));
    gpuErrchk(hipMalloc(&d_teuk_modes, trans_dim2*max_init_len*sizeof(hipDoubleComplex)));
    gpuErrchk(hipMalloc(&d_waveform, max_input_len*sizeof(hipDoubleComplex)));

    //printf("length is %d\n", nit_vals.length);
    temp_t = new fod[max_init_len];
    temp_p = new fod[max_init_len];
    temp_e = new fod[max_init_len];
    temp_Phi_phi = new fod[max_init_len];
    temp_Phi_r = new fod[max_init_len];

    gpuErrchk(hipMalloc(&d_init_t, max_init_len*sizeof(fod)));
    gpuErrchk(hipMalloc(&d_init_p, max_init_len*sizeof(fod)));
    gpuErrchk(hipMalloc(&d_init_e, max_init_len*sizeof(fod)));
    gpuErrchk(hipMalloc(&d_init_Phi_phi, max_init_len*sizeof(fod)));
    gpuErrchk(hipMalloc(&d_init_Phi_r, max_init_len*sizeof(fod)));

    gpuErrchk(hipMalloc(&d_interp_p, sizeof(InterpContainer)));
    gpuErrchk(hipMalloc(&d_interp_e, sizeof(InterpContainer)));
    gpuErrchk(hipMalloc(&d_interp_Phi_phi, sizeof(InterpContainer)));
    gpuErrchk(hipMalloc(&d_interp_Phi_r, sizeof(InterpContainer)));

    h_interp_p = new InterpContainer;
    h_interp_e = new InterpContainer;
    h_interp_Phi_phi = new InterpContainer;
    h_interp_Phi_r = new InterpContainer;

    h_interp_modes = new InterpContainer[num_teuk_modes*2];
    gpuErrchk(hipMalloc(&d_interp_modes, num_teuk_modes*2*sizeof(InterpContainer)));

    create_interp_containers(d_interp_p, h_interp_p, max_init_len);
    create_interp_containers(d_interp_e, h_interp_e, max_init_len);
    create_interp_containers(d_interp_Phi_phi, h_interp_Phi_phi, max_init_len);
    create_interp_containers(d_interp_Phi_r, h_interp_Phi_r, max_init_len);

    create_mode_interp_containers(d_interp_modes, h_interp_modes, max_init_len, num_teuk_modes);

    interp = new InterpClass(num_teuk_modes, max_init_len);


    filter = new FilterContainer;
    gpuErrchk(hipMalloc(&filter->d_mode_keep_inds, num_teuk_modes*sizeof(int)));
    gpuErrchk(hipMalloc(&filter->d_filter_modes_buffer, num_teuk_modes*sizeof(int)));

    gpuErrchk(hipMalloc(&filter->working_modes_all, num_teuk_modes*max_init_len*sizeof(fod)));
    gpuErrchk(hipMalloc(&filter->ind_working_modes_all, num_teuk_modes*max_init_len*sizeof(int)));

    gpuErrchk(hipMalloc(&filter->d_num_modes_kept, sizeof(int)));

    filter->tol = tol_;

}



void FastEMRIWaveforms::run_nn(cmplx *waveform, double M, double mu, double p0, double e0, fod theta, fod phi, int* out_len){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    //gpuErrchk(hipMemcpy(d_Phi_phi, Phi_phi, input_len*sizeof(fod), hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy(d_Phi_r, Phi_r, input_len*sizeof(fod), hipMemcpyHostToDevice));

    double t0 = 0.0;

    hipEventRecord(start);
    NITHolder nit_vals = run_NIT(t0, M, mu, p0, e0, int_err);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("NIT %e\n", milliseconds);

    hipEventRecord(start);
    /*for (int i=0; i<nit_vals.length; i++){
        //printf("[%.18e, %.18e, %.18e, %.18e, %.18e],\n", nit_vals.t_arr[i], nit_vals.p_arr[i], nit_vals.e_arr[i], nit_vals.Phi_phi_arr[i], nit_vals.Phi_r_arr[i]);
        temp_t[i] = (fod)nit_vals.t_arr[i];
        temp_p[i] = (fod)nit_vals.p_arr[i];
        temp_e[i] = (fod)nit_vals.e_arr[i];
        temp_Phi_phi[i] = (fod)nit_vals.Phi_phi_arr[i];
        temp_Phi_r[i] = (fod)nit_vals.Phi_r_arr[i];
    }*/

    // if using double, no need to cast
    temp_t = &nit_vals.t_arr[0];
    temp_p = &nit_vals.p_arr[0];
    temp_e = &nit_vals.e_arr[0];
    temp_Phi_phi = &nit_vals.Phi_phi_arr[0];
    temp_Phi_r = &nit_vals.Phi_r_arr[0];



    gpuErrchk(hipMemcpy(d_init_t, temp_t, nit_vals.length*sizeof(fod), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_init_p, temp_p, nit_vals.length*sizeof(fod), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_init_e, temp_e, nit_vals.length*sizeof(fod), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_init_Phi_phi, temp_Phi_phi, nit_vals.length*sizeof(fod), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_init_Phi_r, temp_Phi_r, nit_vals.length*sizeof(fod), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(h_interp_p->y, temp_p, nit_vals.length*sizeof(fod), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(h_interp_e->y, temp_e, nit_vals.length*sizeof(fod), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(h_interp_Phi_phi->y, temp_Phi_phi, nit_vals.length*sizeof(fod), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(h_interp_Phi_r->y, temp_Phi_r, nit_vals.length*sizeof(fod), hipMemcpyHostToDevice));

    int num_points = std::floor(temp_t[nit_vals.length-1]/delta_t);

    if (num_points > max_input_len) num_points = max_input_len;
    *out_len = num_points;

    //printf("%d num_points, %d max\n", num_points, max_input_len);
    assert(num_points <= max_input_len);
    assert(nit_vals.length <= max_init_len);

    gpuErrchk(hipMemcpy(d_C, d_init_p, nit_vals.length*sizeof(fod), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(&d_C[nit_vals.length], d_init_e, nit_vals.length*sizeof(fod), hipMemcpyDeviceToDevice));

    //ellpe_test(d_C, num_points);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Copy %e\n", milliseconds);

hipEventRecord(start);
  int l,m;
  for (int i=0; i<num_l_m; i+=1){
        l = l_arr[i*num_n];
        m = m_arr[i*num_n];

        Ylms[i] = SpinWeightedSpheroidalHarmonic(l, m, theta, phi);
        Ylms[num_l_m + i] = SpinWeightedSpheroidalHarmonic(l, -m, theta, phi);
        //printf("%d %d, %lf, %lf\n", l , m, Ylms[i].real(), Ylms[i].imag());
  }

  gpuErrchk(hipMemcpy(d_Ylms, Ylms, 2*num_l_m*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Ylms %e\n", milliseconds);

  hipEventRecord(start);
    for (int layer_i=0; layer_i<num_layers; layer_i++){
      run_layer(d_C, d_layers_matrix[layer_i], d_layers_bias[layer_i], dim1[layer_i], dim2[layer_i], nit_vals.length);
    }

    transform_output(d_teuk_modes, d_transform_matrix, d_nn_output_mat, d_C, nit_vals.length, break_index, d_transform_factor_inv, trans_dim2);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("NN and transform: %e\n", milliseconds);

     hipEventRecord(start);
    filter_modes(filter, d_teuk_modes, d_Ylms, d_m, num_teuk_modes, nit_vals.length, num_n, num_l_m);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Filter: %e\n", milliseconds);

     hipEventRecord(start);
    fill_complex_y_vals(d_interp_modes, d_teuk_modes, nit_vals.length, num_teuk_modes, filter);

    interp->setup_interpolate(d_interp_p, d_interp_e, d_interp_Phi_phi, d_interp_Phi_r,
                      d_interp_modes, filter->num_modes_kept, // replaces num_teuk_modes
                           d_init_t, nit_vals.length);

   hipEventRecord(stop);

   hipEventSynchronize(stop);
   hipEventElapsedTime(&milliseconds, start, stop);
   printf("Interpolate: %e\n", milliseconds);

   hipEventRecord(start);
     get_waveform(d_waveform,
                  d_interp_Phi_phi, d_interp_Phi_r, d_interp_modes,
                  d_m, d_n, nit_vals.length, num_points, filter->num_modes_kept, d_Ylms, num_n,
                  delta_t, temp_t, num_l_m, filter);

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Waveform: %e\n", milliseconds);

    //gpuErrchk(hipMemcpy(waveform, d_waveform, num_points*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
}


FastEMRIWaveforms::~FastEMRIWaveforms()
{
    for (int i=0; i<num_layers; i++){
      gpuErrchk(hipFree(d_layers_matrix[i]));
      gpuErrchk(hipFree(d_layers_bias[i]));
    }
    gpuErrchk(hipFree(d_transform_matrix));
    gpuErrchk(hipFree(d_l));
    gpuErrchk(hipFree(d_m));
    gpuErrchk(hipFree(d_n));
    gpuErrchk(hipFree(d_Ylms));
    gpuErrchk(hipFree(d_C));
    gpuErrchk(hipFree(d_nn_output_mat));
    gpuErrchk(hipFree(d_teuk_modes));

    gpuErrchk(hipFree(d_waveform));
    delete[] d_layers_matrix;
    delete[] d_layers_bias;
    delete[] Ylms;

    delete[] temp_t;
    delete[] temp_p;
    delete[] temp_e;
    delete[] temp_Phi_phi;
    delete[] temp_Phi_r;

    destroy_interp_containers(d_interp_p, h_interp_p);
    destroy_interp_containers(d_interp_e, h_interp_e);
    destroy_interp_containers(d_interp_Phi_phi, h_interp_Phi_phi);
    destroy_interp_containers(d_interp_Phi_r, h_interp_Phi_r);

    gpuErrchk(hipFree(d_init_t));
    gpuErrchk(hipFree(d_init_p));
    gpuErrchk(hipFree(d_init_e));
    gpuErrchk(hipFree(d_init_Phi_phi));
    gpuErrchk(hipFree(d_init_Phi_r));

    gpuErrchk(hipFree(d_interp_p));
    gpuErrchk(hipFree(d_interp_e));
    gpuErrchk(hipFree(d_interp_Phi_phi));
    gpuErrchk(hipFree(d_interp_Phi_r));

    delete h_interp_p;
    delete h_interp_e;
    delete h_interp_Phi_phi;
    delete h_interp_Phi_r;

    destroy_mode_interp_containers(d_interp_modes, h_interp_modes, num_teuk_modes);

    gpuErrchk(hipFree(filter->d_filter_modes_buffer));
    gpuErrchk(hipFree(filter->d_mode_keep_inds));
    gpuErrchk(hipFree(filter->working_modes_all));
    gpuErrchk(hipFree(filter->ind_working_modes_all));
    gpuErrchk(hipFree(filter->d_num_modes_kept));

    gpuErrchk(hipFree(d_interp_modes));
    delete[] h_interp_modes;

    delete filter;
    delete interp;
}
