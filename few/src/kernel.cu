#include "hip/hip_runtime.h"
#include <math.h>
#include <random>
#include "global.h"
#include "hipblas.h"

#define NUM_THREADS 256

#define gpuErrchk_here(ans) { gpuAssert_here((ans), __FILE__, __LINE__); }
inline void gpuAssert_here(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

static char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

__global__
void add_bias(fod *C, fod *bias, int input_len, int dim2){

 for (int j = blockIdx.y * blockDim.y + threadIdx.y;
      j < dim2;
      j += blockDim.y * gridDim.y){

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
     i < input_len;
     i += blockDim.x * gridDim.x){

       C[input_len*j + i] = C[input_len*j + i] + bias[j];
  }
}
}


void run_layer(fod *C, fod *layer_weight, fod *layer_bias, int dim1, int dim2, int input_len){
  int m=input_len, k=dim1, n=dim2;
  char * status;
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  fod alpha = 1.0;
  fod beta = 0.0;
  stat = hipblasCreate(&handle);
     if (stat != HIPBLAS_STATUS_SUCCESS) {
             printf ("CUBLAS initialization failed\n");
             exit(0);
         }

  stat = hipblasSgemm(handle,
                         HIPBLAS_OP_N, HIPBLAS_OP_N,
                         m, n, k,
                         &alpha,
                         C, m,
                         layer_weight, k,
                         &beta,
                         C, m);



   status = _cudaGetErrorEnum(stat);
    hipDeviceSynchronize();

    if (stat != HIPBLAS_STATUS_SUCCESS) {
            exit(0);
        }

  int num_blocks = std::ceil((input_len + NUM_THREADS -1)/NUM_THREADS);
  dim3 gridDim(num_blocks, dim2);
  add_bias<<<gridDim, NUM_THREADS>>>(C, layer_bias, input_len, dim2);
  hipDeviceSynchronize();
  gpuErrchk_here(hipGetLastError());
}

__global__
void form_complex_output(hipComplex *complex_output, fod *nn_output, int input_len, int break_index,
                          hipComplex d_transform_factor_inv){
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < input_len;
       i += blockDim.x * gridDim.x){
         for (int ind=0; ind<break_index; ind++){
            complex_output[ind*input_len + i].x = nn_output[ind*input_len + i];
            complex_output[ind*input_len + i].y = nn_output[(break_index+ind)*input_len + i];
            complex_output[ind*input_len + i] = hipCmulf(complex_output[ind*input_len + i], d_transform_factor_inv);
         }
  }
}

void transform_output(hipComplex *d_teuk_modes, hipComplex *d_transform_matrix, hipComplex *d_nn_output_mat, fod *d_C,
                      int input_len, int break_index, hipComplex d_transform_factor_inv,
                      int num_teuk_modes){
  int num_blocks = std::ceil((input_len + NUM_THREADS -1)/NUM_THREADS);
  dim3 gridDim(num_blocks);
  form_complex_output<<<gridDim, NUM_THREADS>>>(d_nn_output_mat, d_C, input_len, break_index, d_transform_factor_inv);
  hipDeviceSynchronize();
  gpuErrchk_here(hipGetLastError());

  int m=input_len, k=break_index, n=num_teuk_modes;
  char * status;
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  hipComplex alpha = make_hipComplex(1.0, 0.0);
  hipComplex beta = make_hipComplex(0.0, 0.0);
  stat = hipblasCreate(&handle);
     if (stat != HIPBLAS_STATUS_SUCCESS) {
             printf ("CUBLAS initialization failed\n");
             exit(0);
         }

  stat = hipblasCgemm(handle,
                         HIPBLAS_OP_N, HIPBLAS_OP_N,
                         m, n, k,
                         &alpha,
                         d_nn_output_mat, m,
                         d_transform_matrix, k,
                         &beta,
                         d_teuk_modes, m);

   status = _cudaGetErrorEnum(stat);
    hipDeviceSynchronize();

    if (stat != HIPBLAS_STATUS_SUCCESS) {
            exit(0);
        }

}
