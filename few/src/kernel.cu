#include "hip/hip_runtime.h"
#include <math.h>
#include <random>
#include "global.h"
#include "hipblas.h"
#include "kernel.hh"

#define NUM_THREADS 256

__device__ __host__ fod LeakyReLU(fod x){
     fod out = 0.0;
     if (x>= 0.0) {out = x;}
     else {out = 0.2*x;}
     return out;
}

__global__
void add_bias_relu(fod *C, fod *bias, int input_len, int dim2){

 for (int j = blockIdx.y * blockDim.y + threadIdx.y;
      j < dim2;
      j += blockDim.y * gridDim.y){

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
     i < input_len;
     i += blockDim.x * gridDim.x){

       C[input_len*j + i] = LeakyReLU(C[input_len*j + i] + bias[j]);
  }
}
}


void run_layer(fod *C, fod *layer_weight, fod *layer_bias, int dim1, int dim2, int input_len){
  int m=input_len, k=dim1, n=dim2;
  char * status;
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  fod alpha = 1.0;
  fod beta = 0.0;
  stat = hipblasCreate(&handle);
     if (stat != HIPBLAS_STATUS_SUCCESS) {
             printf ("CUBLAS initialization failed\n");
             exit(0);
         }

  stat = hipblasSgemm(handle,
                         HIPBLAS_OP_N, HIPBLAS_OP_N,
                         m, n, k,
                         &alpha,
                         C, m,
                         layer_weight, k,
                         &beta,
                         C, m);



   status = _cudaGetErrorEnum(stat);
    hipDeviceSynchronize();

    if (stat != HIPBLAS_STATUS_SUCCESS) {
            exit(0);
        }

  int num_blocks = std::ceil((input_len + NUM_THREADS -1)/NUM_THREADS);
  dim3 gridDim(num_blocks, dim2);
  add_bias_relu<<<gridDim, NUM_THREADS>>>(C, layer_bias, input_len, dim2);
  hipDeviceSynchronize();
  gpuErrchk_here(hipGetLastError());
}

__global__
void form_complex_output(hipComplex *complex_output, fod *nn_output, int input_len, int break_index,
                          hipComplex d_transform_factor_inv){
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < input_len;
       i += blockDim.x * gridDim.x){

   for (int ind = blockIdx.y * blockDim.y + threadIdx.y;
        ind < break_index;
        ind += blockDim.y * gridDim.y){
            complex_output[ind*input_len + i].x = nn_output[ind*input_len + i];
            complex_output[ind*input_len + i].y = nn_output[(break_index+ind)*input_len + i];
            complex_output[ind*input_len + i] = hipCmulf(complex_output[ind*input_len + i], d_transform_factor_inv);
         }
  }
}

void transform_output(hipComplex *d_teuk_modes, hipComplex *d_transform_matrix, hipComplex *d_nn_output_mat, fod *d_C,
                      int input_len, int break_index, hipComplex d_transform_factor_inv,
                      int num_teuk_modes){
  int num_blocks = std::ceil((input_len + NUM_THREADS -1)/NUM_THREADS);
  dim3 gridDim(num_blocks, break_index);
  form_complex_output<<<gridDim, NUM_THREADS>>>(d_nn_output_mat, d_C, input_len, break_index, d_transform_factor_inv);
  hipDeviceSynchronize();
  gpuErrchk_here(hipGetLastError());

  int m=input_len, k=break_index, n=num_teuk_modes;
  char * status;
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  hipComplex alpha = make_hipComplex(1.0, 0.0);
  hipComplex beta = make_hipComplex(0.0, 0.0);
  stat = hipblasCreate(&handle);
     if (stat != HIPBLAS_STATUS_SUCCESS) {
             printf ("CUBLAS initialization failed\n");
             exit(0);
         }

  stat = hipblasCgemm(handle,
                         HIPBLAS_OP_N, HIPBLAS_OP_N,
                         m, n, k,
                         &alpha,
                         d_nn_output_mat, m,
                         d_transform_matrix, k,
                         &beta,
                         d_teuk_modes, m);

   status = _cudaGetErrorEnum(stat);
    hipDeviceSynchronize();

    if (stat != HIPBLAS_STATUS_SUCCESS) {
            exit(0);
        }

}




__host__ __device__ hipComplex complex_exp(hipComplex arg){
  hipComplex res;
  fod s, c;
  fod e = exp(arg.x);
  sincos(arg.y, &s, &c);
  res.x = c * e;
  res.y = s * e;
  return res;
}

__device__
hipComplex get_mode_value(hipComplex teuk_mode, fod Phi_phi, fod Phi_r, int m, int n, hipComplex Ylm){
    hipComplex minus_I = make_hipComplex(0.0, -1.0);
    float phase = m*Phi_phi + n*Phi_r;
    hipComplex out = hipCmulf(hipCmulf(teuk_mode, Ylm), complex_exp(hipCmulf(minus_I, make_hipComplex(phase, 0.0))));
    return out;
}

__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long* address_as_ull =
                              (unsigned long long*)address;
    unsigned long long old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ void atomicAddComplex(hipComplex* a, hipComplex b){
  //transform the addresses of real and imag. parts to double pointers
  double *x = (double*)a;
  double *y = x+1;
  //use atomicAdd for double variables
  atomicAddDouble(x, hipCrealf(b));
  atomicAddDouble(y, hipCimagf(b));
}

__global__
void make_waveform(hipComplex *waveform, hipComplex *teuk_modes, fod *Phi_phi, fod *Phi_r,
              int *m, int *n, int input_len, int num_teuk_modes, hipComplex *Ylms, int num_n){

    hipComplex trans = make_hipComplex(0.0, 0.0);
    hipComplex mode_val, Ylm;
    float Phi_phi_i, Phi_r_i;
    int lm_i;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < input_len;
         i += blockDim.x * gridDim.x){

     Phi_phi_i = Phi_phi[i];
     Phi_r_i = Phi_r[i];
     for (int j = blockIdx.y * blockDim.y + threadIdx.y;
          j < num_teuk_modes;
          j += blockDim.y * gridDim.y){

            lm_i = j / num_n;
            Ylm = Ylms[lm_i];
            //if (i==0) printf("%d %d, %lf + %lfi\n", m[j], n[j], hipCrealf(Ylm), hipCimagf(Ylm));
            mode_val = get_mode_value(teuk_modes[j*input_len + i], Phi_phi_i, Phi_r_i, m[j], n[j], Ylm);
            atomicAddComplex(&waveform[i], mode_val);

    }
  }
}


void get_waveform(hipComplex *d_waveform, hipComplex *d_teuk_modes, fod *d_Phi_phi, fod *d_Phi_r,
              int *d_m, int *d_n, int input_len, int num_teuk_modes, hipComplex *d_Ylms, int num_n){
      int num_blocks = std::ceil((input_len + NUM_THREADS -1)/NUM_THREADS);
      dim3 gridDim(num_blocks, num_teuk_modes); //, num_teuk_modes);
      make_waveform<<<gridDim, NUM_THREADS>>>(d_waveform, d_teuk_modes, d_Phi_phi, d_Phi_r, d_m, d_n, input_len, num_teuk_modes, d_Ylms, num_n);
      hipDeviceSynchronize();
      gpuErrchk_here(hipGetLastError());
}
