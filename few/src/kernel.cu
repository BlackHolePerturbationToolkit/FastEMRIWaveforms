#include "hip/hip_runtime.h"
#include <math.h>
#include <random>
#include "global.h"
#include "hipblas.h"
#include "kernel.hh"
#include "interpolate.hh"

#define NUM_THREADS 256

__device__ __host__ fod LeakyReLU(fod x){
     fod out = 0.0;
     if (x>= 0.0) {out = x;}
     else {out = 0.2*x;}
     return out;
}

__global__
void add_bias_relu(fod *C, fod *bias, int input_len, int dim2){

 for (int j = blockIdx.y * blockDim.y + threadIdx.y;
      j < dim2;
      j += blockDim.y * gridDim.y){

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
     i < input_len;
     i += blockDim.x * gridDim.x){

       C[input_len*j + i] = LeakyReLU(C[input_len*j + i] + bias[j]);
  }
}
}


void run_layer(fod *C, fod *layer_weight, fod *layer_bias, int dim1, int dim2, int input_len){
  int m=input_len, k=dim1, n=dim2;
  char * status;
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  fod alpha = 1.0;
  fod beta = 0.0;
  stat = hipblasCreate(&handle);
     if (stat != HIPBLAS_STATUS_SUCCESS) {
             printf ("CUBLAS initialization failed\n");
             exit(0);
         }

  stat = hipblasSgemm(handle,
                         HIPBLAS_OP_N, HIPBLAS_OP_N,
                         m, n, k,
                         &alpha,
                         C, m,
                         layer_weight, k,
                         &beta,
                         C, m);



   status = _cudaGetErrorEnum(stat);
    hipDeviceSynchronize();

    if (stat != HIPBLAS_STATUS_SUCCESS) {
            exit(0);
        }

  int num_blocks = std::ceil((input_len + NUM_THREADS -1)/NUM_THREADS);
  dim3 gridDim(num_blocks, dim2);
  add_bias_relu<<<gridDim, NUM_THREADS>>>(C, layer_bias, input_len, dim2);
  hipDeviceSynchronize();
  gpuErrchk_here(hipGetLastError());
}

__global__
void form_complex_output(hipComplex *complex_output, fod *nn_output, int input_len, int break_index,
                          hipComplex d_transform_factor_inv){
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < input_len;
       i += blockDim.x * gridDim.x){

   for (int ind = blockIdx.y * blockDim.y + threadIdx.y;
        ind < break_index;
        ind += blockDim.y * gridDim.y){
            complex_output[ind*input_len + i].x = nn_output[ind*input_len + i];
            complex_output[ind*input_len + i].y = nn_output[(break_index+ind)*input_len + i];
            complex_output[ind*input_len + i] = hipCmulf(complex_output[ind*input_len + i], d_transform_factor_inv);
         }
  }
}

void transform_output(hipComplex *d_teuk_modes, hipComplex *d_transform_matrix, hipComplex *d_nn_output_mat, fod *d_C,
                      int input_len, int break_index, hipComplex d_transform_factor_inv,
                      int num_teuk_modes){
  int num_blocks = std::ceil((input_len + NUM_THREADS -1)/NUM_THREADS);
  dim3 gridDim(num_blocks, break_index);
  form_complex_output<<<gridDim, NUM_THREADS>>>(d_nn_output_mat, d_C, input_len, break_index, d_transform_factor_inv);
  hipDeviceSynchronize();
  gpuErrchk_here(hipGetLastError());

  int m=input_len, k=break_index, n=num_teuk_modes;
  char * status;
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  hipComplex alpha = make_hipComplex(1.0, 0.0);
  hipComplex beta = make_hipComplex(0.0, 0.0);
  stat = hipblasCreate(&handle);
     if (stat != HIPBLAS_STATUS_SUCCESS) {
             printf ("CUBLAS initialization failed\n");
             exit(0);
         }

  stat = hipblasCgemm(handle,
                         HIPBLAS_OP_N, HIPBLAS_OP_N,
                         m, n, k,
                         &alpha,
                         d_nn_output_mat, m,
                         d_transform_matrix, k,
                         &beta,
                         d_teuk_modes, m);

   status = _cudaGetErrorEnum(stat);
    hipDeviceSynchronize();

    if (stat != HIPBLAS_STATUS_SUCCESS) {
            exit(0);
        }

}




__host__ __device__ hipComplex complex_exp(hipComplex arg){
  hipComplex res;
  fod s, c;
  fod e = exp(arg.x);
  sincos(arg.y, &s, &c);
  res.x = c * e;
  res.y = s * e;
  return res;
}

__device__
hipComplex get_mode_value(hipComplex teuk_mode, fod Phi_phi, fod Phi_r, int m, int n, hipComplex Ylm){
    hipComplex minus_I = make_hipComplex(0.0, -1.0);
    float phase = m*Phi_phi + n*Phi_r;
    hipComplex out = hipCmulf(hipCmulf(teuk_mode, Ylm), complex_exp(hipCmulf(minus_I, make_hipComplex(phase, 0.0))));
    return out;
}

__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long* address_as_ull =
                              (unsigned long long*)address;
    unsigned long long old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ void atomicAddComplex(hipComplex* a, hipComplex b){
  //transform the addresses of real and imag. parts to double pointers
  double *x = (double*)a;
  double *y = x+1;
  //use atomicAdd for double variables
  atomicAddDouble(x, hipCrealf(b));
  atomicAddDouble(y, hipCimagf(b));
}

__global__
void make_waveform(hipComplex *waveform,
              InterpContainer *Phi_phi_, InterpContainer *Phi_r_, InterpContainer *modes,
              int *m_arr, int *n_arr, int num_teuk_modes, hipComplex *Ylms, int num_n,
              fod delta_t, fod start_t, int old_ind, int start_ind, int end_ind){

    hipComplex trans = make_hipComplex(0.0, 0.0);
    hipComplex mode_val;
    fod Phi_phi_i, Phi_r_i, t, x, x2, x3, mode_val_re, mode_val_im;
    int lm_i;
    __shared__ fod re_y, re_c1, re_c2, re_c3, im_y, im_c1, im_c2, im_c3;
    __shared__ fod pp_y, pp_c1, pp_c2, pp_c3, pr_y, pr_c1, pr_c2, pr_c3;
    __shared__ int m, n;
    __shared__ hipComplex Ylm;

    if (threadIdx.x == 0){

    pp_y = Phi_phi_->y[old_ind]; pp_c1 = Phi_phi_->c1[old_ind]; pp_c2 = Phi_phi_->c2[old_ind]; pp_c3 = Phi_phi_->c3[old_ind];
    pr_y = Phi_phi_->y[old_ind]; pr_c1 = Phi_phi_->c1[old_ind]; pr_c2 = Phi_phi_->c2[old_ind]; pr_c3 = Phi_phi_->c3[old_ind];

    int j = blockIdx.y * blockDim.y + threadIdx.y;

       lm_i = j / num_n;
       Ylm = Ylms[lm_i];

        re_y = modes[2*j].y[old_ind]; re_c1 = modes[2*j].c1[old_ind]; re_c2 = modes[2*j].c2[old_ind]; re_c3 = modes[2*j].c3[old_ind];
        im_y = modes[2*j].y[old_ind]; im_c1 = modes[2*j].c1[old_ind]; im_c2 = modes[2*j].c2[old_ind]; im_c3 = modes[2*j].c3[old_ind];

        m = m_arr[j];
        n = n_arr[j];

    }
    __syncthreads();

    for (int i = start_ind + blockIdx.x * blockDim.x + threadIdx.x;
         i < end_ind;
         i += blockDim.x * gridDim.x){

     t = delta_t*i;
      x = t - start_t;
      x2 = x*x;
      x3 = x*x2;

      Phi_phi_i = pp_y + pp_c1*x + pp_c2*x2  + pp_c3*x3;
      Phi_r_i = pr_y + pr_c1*x + pr_c2*x2  + pr_c3*x3;

        mode_val_re =  re_y + re_c1*x + re_c2*x2  + re_c3*x3;
        mode_val_im = im_y + im_c1*x + im_c2*x2  + im_c3*x3;
        mode_val = make_hipComplex(mode_val_re, mode_val_im);

        //if (i==0) printf("%d %d, %lf + %lfi\n", m[j], n[j], hipCrealf(Ylm), hipCimagf(Ylm));
            mode_val = get_mode_value(mode_val, Phi_phi_i, Phi_r_i, m, n, Ylm);
            atomicAddComplex(&waveform[i], mode_val);

  }
}

void find_start_inds(int start_inds[], int unit_length[], fod *t_arr, fod delta_t, int length, int new_length)
{

  start_inds[0] = 0;
  for (int i = 1;
       i < length;
       i += 1){

          fod t = t_arr[i];

          start_inds[i] = (int)std::ceil(t/delta_t);
          unit_length[i-1] = start_inds[i] - start_inds[i-1];

      }

  start_inds[length -1] = new_length;
  unit_length[length - 2] = start_inds[length -1] - start_inds[length -2];
}

void get_waveform(hipComplex *d_waveform,
              InterpContainer *d_interp_Phi_phi, InterpContainer *d_interp_Phi_r, InterpContainer *d_modes,
              int *d_m, int *d_n, int init_len, int out_len, int num_teuk_modes, hipComplex *d_Ylms, int num_n,
              fod delta_t, fod *h_t){

    int start_inds[init_len];
    int unit_length[init_len-1];

    find_start_inds(start_inds, unit_length, h_t, delta_t, init_len, out_len);

    hipStream_t streams[init_len-1];

    for (int i = 0; i < init_len-2; i++) {
          hipStreamCreate(&streams[i]);
          int num_blocks = std::ceil((unit_length[i] + NUM_THREADS -1)/NUM_THREADS);
          //printf("%d %d %d %d, %d %d\n", i, start_inds[i], unit_length[i], num_blocks, init_len, out_len);
          if (num_blocks == 0) continue;
          dim3 gridDim(num_blocks, num_teuk_modes); //, num_teuk_modes);
          // launch one worker kernel per stream
          make_waveform<<<num_blocks, NUM_THREADS, 0, streams[i]>>>(d_waveform,
                        d_interp_Phi_phi, d_interp_Phi_r, d_modes,
                        d_m, d_n, num_teuk_modes, d_Ylms, num_n,
                        delta_t, h_t[i], i, start_inds[i], start_inds[i+1]);

      }
      hipDeviceSynchronize();
      gpuErrchk(hipGetLastError());
      for (int i = 0; i < init_len-2; i++) {
            hipStreamDestroy(streams[i]);

        }

      /*int num_blocks = std::ceil((input_len + NUM_THREADS -1)/NUM_THREADS);
      dim3 gridDim(num_blocks, num_teuk_modes); //, num_teuk_modes);
      make_waveform<<<gridDim, NUM_THREADS>>>(d_waveform, d_teuk_modes, d_Phi_phi, d_Phi_r, d_m, d_n, input_len, num_teuk_modes, d_Ylms, num_n);
      hipDeviceSynchronize();
      gpuErrchk_here(hipGetLastError());*/
}
