#include "hip/hip_runtime.h"
#include "global.h"
#include "interpolate.hh"

void create_interp_containers(InterpContainer *d_interp, InterpContainer *h_interp, fod *y, int length)
{


  gpuErrchk(hipMalloc(&h_interp->y, length*sizeof(fod)));
  gpuErrchk(hipMalloc(&h_interp->c1, length*sizeof(fod)-1));
  gpuErrchk(hipMalloc(&h_interp->c2, length*sizeof(fod)-1));
  gpuErrchk(hipMalloc(&h_interp->c3, length*sizeof(fod)-1));

  gpuErrchk(hipMemcpy(h_interp->y, y, length*sizeof(fod), hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(d_interp, h_interp, sizeof(InterpContainer), hipMemcpyHostToDevice));

}

void destroy_interp_containers(InterpContainer *d_interp, InterpContainer *h_interp)
{

  gpuErrchk(hipFree(h_interp->y));
  gpuErrchk(hipFree(h_interp->c1));
  gpuErrchk(hipFree(h_interp->c2));
  gpuErrchk(hipFree(h_interp->c3));

}

__device__
void prep_splines(int i, int length, fod *b, fod *ud, fod *diag, fod *ld, fod *x, fod *y){
  fod dx1, dx2, d, slope1, slope2;
  if (i == length - 1){
    dx1 = x[length - 2] - x[length - 3];
    dx2 = x[length - 1] - x[length - 2];
    d = x[length - 1] - x[length - 3];

    slope1 = (y[length - 2] - y[length - 3])/dx1;
    slope2 = (y[length - 1] - y[length - 2])/dx2;

    b[length - 1] = ((dx2*dx2*slope1 +
                             (2*d + dx2)*dx1*slope2) / d);
    diag[length - 1] = dx1;
    ld[length - 1] = d;
    ud[length - 1] = 0.0;

  } else if (i == 0){
      dx1 = x[1] - x[0];
      dx2 = x[2] - x[1];
      d = x[2] - x[0];

      //amp
      slope1 = (y[1] - y[0])/dx1;
      slope2 = (y[2] - y[1])/dx2;

      b[0] = ((dx1 + 2*d) * dx2 * slope1 +
                          dx1*dx1 * slope2) / d;
      diag[0] = dx2;
      ud[0] = d;
      ld[0] = 0.0;

  } else{
    dx1 = x[i] - x[i-1];
    dx2 = x[i+1] - x[i];

    //amp
    slope1 = (y[i] - y[i-1])/dx1;
    slope2 = (y[i+1] - y[i])/dx2;

    b[i] = 3.0* (dx2*slope1 + dx1*slope2);
    diag[i] = 2*(dx1 + dx2);
    ud[i] = dx1;
    ld[i] = dx2;
  }
}


__device__
void fill_B(fod *p, fod *e, fod *Phi_phi, fod *Phi_r,
                 fod *B, fod *t_arr, fod *upper_diag, fod *diag, fod *lower_diag, int length, int i){
    int lead_ind;

    // p
    lead_ind = 0*length;
    prep_splines(i, length, &B[lead_ind], &upper_diag[lead_ind], &diag[lead_ind], &lower_diag[lead_ind], t_arr, p);

    // e
    lead_ind = 1*length;
    prep_splines(i, length, &B[lead_ind], &upper_diag[lead_ind], &diag[lead_ind], &lower_diag[lead_ind], t_arr, e);

    // Phi_phi
    lead_ind = 2*length;
    prep_splines(i, length, &B[lead_ind], &upper_diag[lead_ind], &diag[lead_ind], &lower_diag[lead_ind], t_arr, Phi_phi);

    // Phi_r
    lead_ind = 3*length;
    prep_splines(i, length, &B[lead_ind], &upper_diag[lead_ind], &diag[lead_ind], &lower_diag[lead_ind], t_arr, Phi_r);

}

__global__
void fill_B_wrap(InterpContainer *p_, InterpContainer *e_, InterpContainer *Phi_phi_, InterpContainer *Phi_r_,
                      fod *t_arr, fod *B, fod *upper_diag, fod *diag, fod *lower_diag, int length){

        fod *p = p_->y;
        fod *e = e_->y;
        fod *Phi_phi = Phi_phi_->y;
        fod *Phi_r = Phi_r_->y;

       for (int i = blockIdx.x * blockDim.x + threadIdx.x;
            i < length;
            i += blockDim.x * gridDim.x){

              fill_B(p, e, Phi_phi, Phi_r, B, t_arr, upper_diag, diag, lower_diag, length, i);

}
}

void fit_constants_serial_wrap(int m, int n, fod *a, fod *b, fod *c, fod *d_in){

  void *pBuffer;
  hipsparseStatus_t stat;
  hipsparseHandle_t handle;

  size_t bufferSizeInBytes;

  CUSPARSE_CALL(hipsparseCreate(&handle));
  CUSPARSE_CALL( hipsparseSgtsv2StridedBatch_bufferSizeExt(handle, m, a, b, c, d_in, n, m, &bufferSizeInBytes));
  gpuErrchk(hipMalloc(&pBuffer, bufferSizeInBytes));

    CUSPARSE_CALL(hipsparseSgtsv2StridedBatch(handle,
                                              m,
                                              a, // dl
                                              b, //diag
                                              c, // du
                                              d_in,
                                              n,
                                              m,
                                              pBuffer));


CUSPARSE_CALL(hipsparseDestroy(handle));
gpuErrchk(hipFree(pBuffer));
}

__device__
void fill_coefficients(int i, int length, fod *dydx, fod dx, fod *y, fod *coeff1, fod *coeff2, fod *coeff3){
  fod slope, t, dydx_i;

  slope = (y[i+1] - y[i])/dx;

  dydx_i = dydx[i];

  t = (dydx_i + dydx[i+1] - 2*slope)/dx;

  coeff1[i] = dydx_i;
  coeff2[i] = (slope - dydx_i) / dx - t;
  coeff3[i] = t/dx;
}


__device__
void set_spline_constants(InterpContainer *p_, InterpContainer *e_, InterpContainer *Phi_phi_, InterpContainer *Phi_r_,
                          fod *B, int length, int i, fod dt){

  int lead_ind;

  // p
  lead_ind = 0*length;
  fill_coefficients(i, length, &B[lead_ind], dt, p_->y, p_->c1, p_->c2, p_->c3);

  // e
  lead_ind = 1*length;
  fill_coefficients(i, length, &B[lead_ind], dt, e_->y, e_->c1, e_->c2, e_->c3);

  // Phi_phi
  lead_ind = 2*length;
  fill_coefficients(i, length, &B[lead_ind], dt, Phi_phi_->y, Phi_phi_->c1, Phi_phi_->c2, Phi_phi_->c3);

  // Phi_r
  lead_ind = 3*length;
  fill_coefficients(i, length, &B[lead_ind], dt, Phi_r_->y, Phi_r_->c1, Phi_r_->c2, Phi_r_->c3);

}

__global__
void set_spline_constants_wrap(InterpContainer *p_, InterpContainer *e_, InterpContainer *Phi_phi_, InterpContainer *Phi_r_,
                               fod *B, int length, fod *t_arr){

    fod dt;

        for (int i = blockIdx.x * blockDim.x + threadIdx.x;
             i < length-1;
             i += blockDim.x * gridDim.x){

              dt = t_arr[i + 1] - t_arr[i];

               set_spline_constants(p_, e_, Phi_phi_, Phi_r_, B, length, i, dt);
}
}


void setup_interpolate(InterpContainer *d_interp_p, InterpContainer *d_interp_e, InterpContainer *d_interp_Phi_phi, InterpContainer *d_interp_Phi_r,
                       fod *d_t, int length)
{

  int num_pars = 4;
  fod *upper_diag, *lower_diag, *diag, *B;

  gpuErrchk(hipMalloc(&upper_diag, num_pars*length*sizeof(fod)));
  gpuErrchk(hipMalloc(&lower_diag, num_pars*length*sizeof(fod)));
  gpuErrchk(hipMalloc(&diag, num_pars*length*sizeof(fod)));
  gpuErrchk(hipMalloc(&B, num_pars*length*sizeof(fod)));

  int NUM_THREADS = 256;
  int num_blocks = std::ceil((length + NUM_THREADS -1)/NUM_THREADS);
  dim3 gridDim(num_blocks); //, num_teuk_modes);
  fill_B_wrap<<<gridDim, NUM_THREADS>>>(d_interp_p, d_interp_e, d_interp_Phi_phi, d_interp_Phi_r,
                        d_t, B, upper_diag, diag, lower_diag, length);
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());

  fit_constants_serial_wrap(length, num_pars, lower_diag, diag, upper_diag, B);

  set_spline_constants_wrap<<<gridDim, NUM_THREADS>>>(d_interp_p, d_interp_e, d_interp_Phi_phi, d_interp_Phi_r,
                                 B, length, d_t);
 hipDeviceSynchronize();
 gpuErrchk(hipGetLastError());

  gpuErrchk(hipFree(upper_diag));
  gpuErrchk(hipFree(lower_diag));
  gpuErrchk(hipFree(diag));
  gpuErrchk(hipFree(B));

}
