#include "hip/hip_runtime.h"
// Interpolate and sum modes for an EMRI waveform

// Copyright (C) 2020 Michael L. Katz, Alvin J.K. Chua, Niels Warburton, Scott A. Hughes
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <https://www.gnu.org/licenses/>.

#include "global.h"
#include "interpolate.hh"
#include <stdio.h>

// adjust imports based on CUDA or not
#ifdef __HIPCC__
#include "hipsparse.h"
#else
#include "lapacke.h"
#endif
#ifdef __USE_OMP__
#include "omp.h"
#endif


#ifdef __HIPCC__
#define MAX_MODES_BLOCK 450
#else
#define MAX_MODES_BLOCK 5000
#endif

#define NUM_TERMS 4

// fills the coefficients of the cubic spline
// according to scipy Cubic Spline
CUDA_CALLABLE_MEMBER
void fill_coefficients(int i, int length, double *dydx, double dx, double *y, double *coeff1, double *coeff2, double *coeff3)
{
  double slope, t, dydx_i;

  slope = (y[i+1] - y[i])/dx;

  dydx_i = dydx[i];

  t = (dydx_i + dydx[i+1] - 2*slope)/dx;

  coeff1[i] = dydx_i;
  coeff2[i] = (slope - dydx_i) / dx - t;
  coeff3[i] = t/dx;
}

// fills the banded matrix that will be solved for spline coefficients
// according to scipy Cubic Spline
  // this performs a not-a-knot spline
CUDA_CALLABLE_MEMBER
void prep_splines(int i, int length, double *b, double *ud, double *diag, double *ld, double *x, double *y)
{
    double dx1, dx2, d, slope1, slope2;

    // this performs a not-a-knot spline
    // need to adjust for ends of the splines
    if (i == length - 1)
    {
        dx1 = x[length - 2] - x[length - 3];
        dx2 = x[length - 1] - x[length - 2];
        d = x[length - 1] - x[length - 3];

        slope1 = (y[length - 2] - y[length - 3])/dx1;
        slope2 = (y[length - 1] - y[length - 2])/dx2;

        b[length - 1] = ((dx2*dx2*slope1 +
                                 (2*d + dx2)*dx1*slope2) / d);
        diag[length - 1] = dx1;
        ld[length - 1] = d;
        ud[length - 1] = 0.0;

    }

    else if (i == 0)
    {
        dx1 = x[1] - x[0];
        dx2 = x[2] - x[1];
        d = x[2] - x[0];

        slope1 = (y[1] - y[0])/dx1;
        slope2 = (y[2] - y[1])/dx2;

        b[0] = ((dx1 + 2*d) * dx2 * slope1 +
                          dx1*dx1 * slope2) / d;
        diag[0] = dx2;
        ud[0] = d;
        ld[0] = 0.0;

    }

    else
    {
        dx1 = x[i] - x[i-1];
        dx2 = x[i+1] - x[i];

        slope1 = (y[i] - y[i-1])/dx1;
        slope2 = (y[i+1] - y[i])/dx2;

        b[i] = 3.0* (dx2*slope1 + dx1*slope2);
        diag[i] = 2*(dx1 + dx2);
        ud[i] = dx1;
        ld[i] = dx2;
    }
}


// wrapper to fill the banded matrix that will be solved for spline coefficients
// according to scipy Cubic Spline
CUDA_KERNEL
void fill_B(double *t_arr, double *y_all, double *B, double *upper_diag, double *diag, double *lower_diag,
                      int ninterps, int length)
{

    #ifdef __HIPCC__

    int start1 = blockIdx.y*blockDim.y + threadIdx.y;
    int end1 = ninterps;
    int diff1 = blockDim.y*gridDim.y;

    int start2 = blockIdx.x*blockDim.x + threadIdx.x;
    int end2 = length;
    int diff2 = blockDim.x * gridDim.x;
    #else

    int start1 = 0;
    int end1 = ninterps;
    int diff1 = 1;

    int start2 = 0;
    int end2 = length;
    int diff2 = 1;

    #pragma omp parallel for
    #endif
    for (int interp_i= start1;
         interp_i<end1; // 2 for re and im
         interp_i+= diff1)
         {

       for (int i = start2;
            i < end2;
            i += diff2)
            {

                int lead_ind = interp_i*length;
                prep_splines(i, length, &B[lead_ind], &upper_diag[lead_ind], &diag[lead_ind], &lower_diag[lead_ind], &t_arr[lead_ind], &y_all[interp_i*length]);
            }
        }
}


// wrapper to set spline coefficients
// according to scipy Cubic Spline
CUDA_KERNEL
void set_spline_constants(double *t_arr, double *interp_array, double *B,
                      int ninterps, int length)
{

    double dt;
    InterpContainer mode_vals;

    #ifdef __HIPCC__
    int start1 = blockIdx.y*blockDim.y + threadIdx.y;
    int end1 = ninterps;
    int diff1 = blockDim.y*gridDim.y;

    int start2 = blockIdx.x*blockDim.x + threadIdx.x;
    int end2 = length - 1;
    int diff2 = blockDim.x * gridDim.x;
    #else

    int start1 = 0;
    int end1 = ninterps;
    int diff1 = 1;

    int start2 = 0;
    int end2 = length - 1;
    int diff2 = 1;

    #pragma omp parallel for
    #endif

    for (int interp_i= start1;
         interp_i<end1; // 2 for re and im
         interp_i+= diff1)
         {

       for (int i = start2;
            i < end2;
            i += diff2)
            {

              dt = t_arr[interp_i * length + i + 1] - t_arr[interp_i * length + i];

              int lead_ind = interp_i*length;
              fill_coefficients(i, length, &B[lead_ind], dt,
                                &interp_array[0 * ninterps * length + lead_ind],
                                &interp_array[1 * ninterps * length + lead_ind],
                                &interp_array[2 * ninterps * length + lead_ind],
                                &interp_array[3 * ninterps * length + lead_ind]);

             }
        }
}


// wrapper for cusparse solution for coefficients from banded matrix
void fit_wrap(int m, int n, double *a, double *b, double *c, double *d_in)
{
    #ifdef __HIPCC__
    size_t bufferSizeInBytes;

    hipsparseHandle_t handle;
    void *pBuffer;

    CUSPARSE_CALL(hipsparseCreate(&handle));
    CUSPARSE_CALL( hipsparseDgtsv2StridedBatch_bufferSizeExt(handle, m, a, b, c, d_in, n, m, &bufferSizeInBytes));
    gpuErrchk(hipMalloc(&pBuffer, bufferSizeInBytes));

    // solve banded matrix problem
    CUSPARSE_CALL(hipsparseDgtsv2StridedBatch(handle,
                                              m,
                                              a, // dl
                                              b, //diag
                                              c, // du
                                              d_in,
                                              n,
                                              m,
                                              pBuffer));

  CUSPARSE_CALL(hipsparseDestroy(handle));
  gpuErrchk(hipFree(pBuffer));

  #else

    // use lapack on CPU
    #ifdef __USE_OMP__
    #pragma omp parallel for
    #endif
    for (int j = 0;
         j < n;
         j += 1)
         {
               int info = LAPACKE_dgtsv(LAPACK_COL_MAJOR, m, 1, &a[j*m + 1], &b[j*m], &c[j*m], &d_in[j*m], m);
         }

  #endif
}

// interpolate many y arrays (interp_array) with a singular x array (t_arr)
// see python documentation for shape necessary for this to be done
void interpolate_arrays(double *t_arr, double *interp_array, int ninterps, int length, double *B, double *upper_diag, double *diag, double *lower_diag)
{

    // need to fill the banded matrix
    // solve it
    // fill the coefficient arrays
    // do that below on GPU or CPU

  #ifdef __HIPCC__
  int NUM_THREADS = 64;
  int num_blocks = std::ceil((length + NUM_THREADS -1)/NUM_THREADS);
  dim3 gridDim(num_blocks); //, num_teuk_modes);
  fill_B<<<gridDim, NUM_THREADS>>>(t_arr, interp_array, B, upper_diag, diag, lower_diag, ninterps, length);
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());

  fit_wrap(length, ninterps, lower_diag, diag, upper_diag, B);

  set_spline_constants<<<gridDim, NUM_THREADS>>>(t_arr, interp_array, B,
                                 ninterps, length);
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());

  #else

  fill_B(t_arr, interp_array, B, upper_diag, diag, lower_diag, ninterps, length);

  fit_wrap(length, ninterps, lower_diag, diag, upper_diag, B);

  set_spline_constants(t_arr, interp_array, B,
                                 ninterps, length);

  #endif

}

/////////////////////////////////
/////////
/////////  MODE SUMMATION
/////////
/////////////////////////////////


// build mode value with specific phase and amplitude values; mode indexes; and spherical harmonics
CUDA_CALLABLE_MEMBER
cmplx get_mode_value(cmplx teuk_mode, fod Phi_phi, fod Phi_r, int m, int n, cmplx Ylm){
    cmplx minus_I(0.0, -1.0);
    fod phase = m*Phi_phi + n*Phi_r;
    cmplx out = (teuk_mode*Ylm)*gcmplx::exp(minus_I*phase);
    return out;
}

// Add functionality for proper summation in the kernel
#ifdef __HIPCC__
__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long* address_as_ull =
                              (unsigned long long*)address;
    unsigned long long old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

// Add functionality for proper summation in the kernel
__device__ void atomicAddComplex(cmplx* a, cmplx b){
  //transform the addresses of real and imag. parts to double pointers
  double *x = (double*)a;
  double *y = x+1;
  //use atomicAdd for double variables
  atomicAddDouble(x, b.real());
  atomicAddDouble(y, b.imag());
}

#endif


// make a waveform in parallel
// this uses an efficient summation by loading mode information into shared memory
// shared memory is leveraged heavily
CUDA_KERNEL
void make_waveform(cmplx *waveform,
             double *interp_array,
              int *m_arr_in, int *n_arr_in, int num_teuk_modes, cmplx *Ylms_in,
              double delta_t, double start_t, int old_ind, int start_ind, int end_ind, int init_length){

    int num_pars = 2;
    cmplx trans(0.0, 0.0);
    cmplx trans2(0.0, 0.0);

    cmplx complexI(0.0, 1.0);
    cmplx mode_val;
    cmplx trans_plus_m(0.0, 0.0), trans_minus_m(0.0, 0.0);
    double Phi_phi_i, Phi_r_i, t, x, x2, x3, mode_val_re, mode_val_im;
    int lm_i, num_teuk_here;
    double re_y, re_c1, re_c2, re_c3, im_y, im_c1, im_c2, im_c3;
     CUDA_SHARED double pp_y, pp_c1, pp_c2, pp_c3, pr_y, pr_c1, pr_c2, pr_c3;

     // declare all the shared memory
     // MAX_MODES_BLOCK is fixed based on shared memory
     CUDA_SHARED cmplx Ylms[2*MAX_MODES_BLOCK];
     CUDA_SHARED double mode_re_y[MAX_MODES_BLOCK];
     CUDA_SHARED double mode_re_c1[MAX_MODES_BLOCK];
     CUDA_SHARED double mode_re_c2[MAX_MODES_BLOCK];
     CUDA_SHARED double mode_re_c3[MAX_MODES_BLOCK];

     CUDA_SHARED double mode_im_y[MAX_MODES_BLOCK];
     CUDA_SHARED double mode_im_c1[MAX_MODES_BLOCK];
     CUDA_SHARED double mode_im_c2[MAX_MODES_BLOCK];
     CUDA_SHARED double mode_im_c3[MAX_MODES_BLOCK];

     CUDA_SHARED int m_arr[MAX_MODES_BLOCK];
     CUDA_SHARED int n_arr[MAX_MODES_BLOCK];

     // number of splines
     int num_base = init_length * (2 * num_teuk_modes + num_pars);

     CUDA_SYNC_THREADS;

     #ifdef __HIPCC__

     if ((threadIdx.x == 0)){
     #else
     if (true){
     #endif

        // fill phase values. These will be same for all modes
         int ind_Phi_phi = old_ind*(2*num_teuk_modes+num_pars) + num_teuk_modes*2 + 0;
         int ind_Phi_r = old_ind*(2*num_teuk_modes+num_pars) + num_teuk_modes*2 + 1;

         pp_y = interp_array[0 * num_base + ind_Phi_phi]; pp_c1 = interp_array[1 * num_base + ind_Phi_phi];
         pp_c2= interp_array[2 * num_base + ind_Phi_phi];  pp_c3 = interp_array[3 * num_base + ind_Phi_phi];

         pr_y = interp_array[0 * num_base + ind_Phi_r]; pr_c1 = interp_array[1 * num_base + ind_Phi_r];
         pr_c2= interp_array[2 * num_base + ind_Phi_r];  pr_c3 = interp_array[3 * num_base + ind_Phi_r];
     }

     CUDA_SYNC_THREADS;

     int m, n, actual_mode_index;
     cmplx Ylm_plus_m, Ylm_minus_m;

     int num_breaks = (num_teuk_modes / MAX_MODES_BLOCK) + 1;

     // this does a special loop to fill mode information into shared memory in chunks
     for (int block_y=0; block_y<num_breaks; block_y+=1){
    num_teuk_here = (((block_y + 1)*MAX_MODES_BLOCK) <= num_teuk_modes) ? MAX_MODES_BLOCK : num_teuk_modes - (block_y*MAX_MODES_BLOCK);

    int init_ind = block_y*MAX_MODES_BLOCK;


    #ifdef __HIPCC__

    int start = threadIdx.x;
    int end = num_teuk_here;
    int diff = blockDim.x;

    #else

    int start = 0;
    int end = num_teuk_here;
    int diff = 1;
    #ifdef __USE_OMP__
    #pragma omp parallel for
    #endif // __USE_OMP__
    #endif
    for (int i=start; i<end; i+=diff)
    {

        // fill mode values and Ylms
        int ind_re = old_ind*(2*num_teuk_modes+num_pars) + (init_ind + i);
        int ind_im = old_ind*(2*num_teuk_modes+num_pars)  + num_teuk_modes + (init_ind + i);
        mode_re_y[i] = interp_array[0 * num_base + ind_re]; mode_re_c1[i] = interp_array[1 * num_base + ind_re];
        mode_re_c2[i] = interp_array[2 * num_base + ind_re]; mode_re_c3[i] = interp_array[3 * num_base + ind_re];

        mode_im_y[i] = interp_array[0 * num_base + ind_im]; mode_im_c1[i] = interp_array[1 * num_base + ind_im];
        mode_im_c2[i] = interp_array[2 * num_base + ind_im]; mode_im_c3[i] = interp_array[3 * num_base + ind_im];

        m_arr[i] = m_arr_in[init_ind + i];
        n_arr[i] = n_arr_in[init_ind + i];
        Ylms[2*i] = Ylms_in[(init_ind + i)];
        Ylms[2*i + 1] = Ylms_in[num_teuk_modes + (init_ind + i)];
    }

    CUDA_SYNC_THREADS;

    #ifdef __HIPCC__

    start = start_ind + blockIdx.x * blockDim.x + threadIdx.x;
    end = end_ind;
    diff = blockDim.x * gridDim.x;

    #else

    start = start_ind;
    end = end_ind;
    diff = 1;

    #endif
    #ifdef __HIPCC__
    #else
    #ifdef __USE_OMP__
    #pragma omp parallel for
    #endif // __USE_OMP__
    #endif // __HIPCC__

    // start and end is the start and end of points in this interpolation window
    for (int i = start;
         i < end;
         i += diff){

     trans2 = 0.0 + 0.0*complexI;

     trans = 0.0 + 0.0*complexI;

     // determine interpolation information
     t = delta_t*i;
      x = t - start_t;
      x2 = x*x;
      x3 = x*x2;

      // get phases at this timestep
      Phi_phi_i = pp_y + pp_c1*x + pp_c2*x2  + pp_c3*x3;
      Phi_r_i = pr_y + pr_c1*x + pr_c2*x2  + pr_c3*x3;

      // calculate all modes at this timestep
        for (int j=0; j<num_teuk_here; j+=1){

            Ylm_plus_m = Ylms[2*j];

             m = m_arr[j];
             n = n_arr[j];

            mode_val_re =  mode_re_y[j] + mode_re_c1[j]*x + mode_re_c2[j]*x2  + mode_re_c3[j]*x3;
            mode_val_im = mode_im_y[j] + mode_im_c1[j]*x + mode_im_c2[j]*x2  + mode_im_c3[j]*x3;
            mode_val = mode_val_re + complexI*mode_val_im;

                trans_plus_m = get_mode_value(mode_val, Phi_phi_i, Phi_r_i, m, n, Ylm_plus_m);

                // minus m if m > 0
                // mode values for +/- m are taking care of when applying
                //specific mode selection by setting ylms to zero for the opposites
                if (m != 0)
                {

                    Ylm_minus_m = Ylms[2*j + 1];
                    trans_minus_m = get_mode_value(gcmplx::conj(mode_val), Phi_phi_i, Phi_r_i, -m, -n, Ylm_minus_m);

                } else trans_minus_m = 0.0 + 0.0*complexI;

                trans = trans + trans_minus_m + trans_plus_m;
        }

        // fill waveform
        #ifdef __HIPCC__
        atomicAddComplex(&waveform[i], trans);
        #else
        waveform[i] += trans;
        #endif
    }
    CUDA_SYNC_THREADS;
}
}


// with uneven spacing in t in the sparse arrays, need to determine which timesteps the dense arrays fall into
// for interpolation
// effectively the boundaries and length of each interpolation segment of the dense array in the sparse array
void find_start_inds(int start_inds[], int unit_length[], double *t_arr, double delta_t, int *length, int new_length)
{

    double T = (new_length - 1) * delta_t;
  start_inds[0] = 0;
  int i = 1;
  for (i = 1;
       i < *length;
       i += 1){

          double t = t_arr[i];

          // adjust for waveforms that hit the end of the trajectory
          if (t < T){
              start_inds[i] = (int)std::ceil(t/delta_t);
              unit_length[i-1] = start_inds[i] - start_inds[i-1];
          } else {
            start_inds[i] = new_length;
            unit_length[i-1] = new_length - start_inds[i-1];
            break;
        }

      }

  // fixes for not using certain segments for the interpolation
  *length = i + 1;
}

// function for building interpolated EMRI waveform from python
void get_waveform(cmplx *d_waveform, double *interp_array,
              int *d_m, int *d_n, int init_len, int out_len, int num_teuk_modes, cmplx *d_Ylms,
              double delta_t, double *h_t){

    // arrays for determining spline windows for new arrays
    int start_inds[init_len];
    int unit_length[init_len-1];

    int number_of_old_spline_points = init_len;

    // find the spline window information based on equally spaced new array
    find_start_inds(start_inds, unit_length, h_t, delta_t, &number_of_old_spline_points, out_len);

    #ifdef __HIPCC__

    // prepare streams for CUDA
    int NUM_THREADS = 256;
    hipStream_t streams[number_of_old_spline_points-1];
    int num_breaks = num_teuk_modes/MAX_MODES_BLOCK;

    #endif

    #ifdef __USE_OMP__
    #pragma omp parallel for
    #endif
    for (int i = 0; i < number_of_old_spline_points-1; i++) {
          #ifdef __HIPCC__

          // create and execute with streams
          hipStreamCreate(&streams[i]);
          int num_blocks = std::ceil((unit_length[i] + NUM_THREADS -1)/NUM_THREADS);

          // sometimes a spline interval will have zero points
          if (num_blocks <= 0) continue;

          dim3 gridDim(num_blocks, 1);

          // launch one worker kernel per stream
          make_waveform<<<gridDim, NUM_THREADS, 0, streams[i]>>>(d_waveform,
                        interp_array,
                        d_m, d_n, num_teuk_modes, d_Ylms,
                        delta_t, h_t[i], i, start_inds[i], start_inds[i+1], init_len);
         #else

         // CPU waveform generation
         make_waveform(d_waveform,
                       interp_array,
                       d_m, d_n, num_teuk_modes, d_Ylms,
                       delta_t, h_t[i], i, start_inds[i], start_inds[i+1], init_len);
         #endif

      }

      //synchronize after all streams finish
      #ifdef __HIPCC__
      hipDeviceSynchronize();
      gpuErrchk(hipGetLastError());

      #ifdef __USE_OMP__
      #pragma omp parallel for
      #endif
      for (int i = 0; i < number_of_old_spline_points-1; i++) {
            //destroy the streams
            hipStreamDestroy(streams[i]);
        }
      #endif
}



// build mode value with specific phase and amplitude values; mode indexes; and spherical harmonics
CUDA_CALLABLE_MEMBER
cmplx get_mode_value_generic(cmplx teuk_mode, fod Phi_phi, fod Phi_theta, fod Phi_r, int m, int k, int n){
    cmplx minus_I(0.0, -1.0);
    fod phase = m * Phi_phi + k * Phi_theta + n * Phi_r;
    cmplx out = teuk_mode * gcmplx::exp(minus_I*phase);
    return out;
}

// make a waveform in parallel
// this uses an efficient summation by loading mode information into shared memory
// shared memory is leveraged heavily
#define MAX_SPLINE_POINTS 210
CUDA_KERNEL
void make_generic_kerr_waveform(cmplx *waveform,
             double *interp_array,
              int *m_arr_in, int *k_arr_in, int *n_arr_in, int num_teuk_modes,
              double delta_t, double *old_time_arr, int init_length, int data_length, int *interval_inds, bool separate_modes){

    int num_pars = 3;
  

    cmplx complexI(0.0, 1.0);
    double re_y, re_c1, re_c2, re_c3, im_y, im_c1, im_c2, im_c3;
     CUDA_SHARED double pp_y, pp_c1, pp_c2, pp_c3, pr_y, pr_c1, pr_c2, pr_c3;

     // declare all the shared memory
     // MAX_MODES_BLOCK is fixed based on shared memory
     CUDA_SHARED double old_time[MAX_SPLINE_POINTS];

     CUDA_SHARED double R_mode_re_y[MAX_SPLINE_POINTS];
     CUDA_SHARED double R_mode_re_c1[MAX_SPLINE_POINTS];
     CUDA_SHARED double R_mode_re_c2[MAX_SPLINE_POINTS];
     CUDA_SHARED double R_mode_re_c3[MAX_SPLINE_POINTS];

     CUDA_SHARED double R_mode_im_y[MAX_SPLINE_POINTS];
     CUDA_SHARED double R_mode_im_c1[MAX_SPLINE_POINTS];
     CUDA_SHARED double R_mode_im_c2[MAX_SPLINE_POINTS];
     CUDA_SHARED double R_mode_im_c3[MAX_SPLINE_POINTS];

     CUDA_SHARED double L_mode_re_y[MAX_SPLINE_POINTS];
     CUDA_SHARED double L_mode_re_c1[MAX_SPLINE_POINTS];
     CUDA_SHARED double L_mode_re_c2[MAX_SPLINE_POINTS];
     CUDA_SHARED double L_mode_re_c3[MAX_SPLINE_POINTS];

     CUDA_SHARED double L_mode_im_y[MAX_SPLINE_POINTS];
     CUDA_SHARED double L_mode_im_c1[MAX_SPLINE_POINTS];
     CUDA_SHARED double L_mode_im_c2[MAX_SPLINE_POINTS];
     CUDA_SHARED double L_mode_im_c3[MAX_SPLINE_POINTS];

     CUDA_SHARED double Phi_phi_y[MAX_SPLINE_POINTS];
     CUDA_SHARED double Phi_phi_c1[MAX_SPLINE_POINTS];
     CUDA_SHARED double Phi_phi_c2[MAX_SPLINE_POINTS];
     CUDA_SHARED double Phi_phi_c3[MAX_SPLINE_POINTS];

     CUDA_SHARED double Phi_theta_y[MAX_SPLINE_POINTS];
     CUDA_SHARED double Phi_theta_c1[MAX_SPLINE_POINTS];
     CUDA_SHARED double Phi_theta_c2[MAX_SPLINE_POINTS];
     CUDA_SHARED double Phi_theta_c3[MAX_SPLINE_POINTS];

     CUDA_SHARED double Phi_r_y[MAX_SPLINE_POINTS];
     CUDA_SHARED double Phi_r_c1[MAX_SPLINE_POINTS];
     CUDA_SHARED double Phi_r_c2[MAX_SPLINE_POINTS];
     CUDA_SHARED double Phi_r_c3[MAX_SPLINE_POINTS];
       
     // number of splines
     int num_base = (4 * num_teuk_modes + num_pars) * init_length;

    #ifdef __HIPCC__

    int start2 = blockIdx.y;
    int diff2 = gridDim.y;

    #else

    int start2 = 0;
    int diff2 = 1;
    #ifdef __USE_OMP__
    #pragma omp parallel for
    #endif // __USE_OMP__
    #endif
    for (int mode_i = start2; mode_i < num_teuk_modes; mode_i += diff2) 
    {
      
      int m = m_arr_in[mode_i];
      int k = k_arr_in[mode_i];
      int n = n_arr_in[mode_i];

      CUDA_SYNC_THREADS;
     
      #ifdef __HIPCC__

      int start = threadIdx.x;
      int diff = blockDim.x;

      #else

      int start = 0;
      int diff = 1;
      #ifdef __USE_OMP__
      #pragma omp parallel for
      #endif // __USE_OMP__
      #endif
      for (int i = start; i < init_length; i += diff)
      {
          old_time[i] = old_time_arr[i];

          int y_ind = 0 * num_base + mode_i * init_length + i;
          int c1_ind = 1 * num_base + mode_i * init_length + i;
          int c2_ind = 2 * num_base + mode_i * init_length + i;
          int c3_ind = 3 * num_base + mode_i * init_length + i;

          R_mode_re_y[i] = interp_array[y_ind];
          R_mode_re_c1[i] = interp_array[c1_ind];
          R_mode_re_c2[i] = interp_array[c2_ind];
          R_mode_re_c3[i] = interp_array[c3_ind];

          y_ind = 0 * num_base + (num_teuk_modes + mode_i) * init_length + i;
          c1_ind = 1 * num_base + (num_teuk_modes + mode_i) * init_length + i;
          c2_ind = 2 * num_base + (num_teuk_modes + mode_i) * init_length + i;
          c3_ind = 3 * num_base + (num_teuk_modes + mode_i) * init_length + i;

          R_mode_im_y[i] = interp_array[y_ind];
          R_mode_im_c1[i] = interp_array[c1_ind];
          R_mode_im_c2[i] = interp_array[c2_ind];
          R_mode_im_c3[i] = interp_array[c3_ind];

          y_ind = 0 * num_base + (2 * num_teuk_modes + mode_i) * init_length + i;
          c1_ind = 1 * num_base + (2 * num_teuk_modes + mode_i) * init_length + i;
          c2_ind = 2 * num_base + (2 * num_teuk_modes + mode_i) * init_length + i;
          c3_ind = 3 * num_base + (2 * num_teuk_modes + mode_i) * init_length + i;

          L_mode_re_y[i] = interp_array[y_ind];
          L_mode_re_c1[i] = interp_array[c1_ind];
          L_mode_re_c2[i] = interp_array[c2_ind];
          L_mode_re_c3[i] = interp_array[c3_ind];

          y_ind = 0 * num_base + (3 * num_teuk_modes + mode_i) * init_length + i;
          c1_ind = 1 * num_base + (3 * num_teuk_modes + mode_i) * init_length + i;
          c2_ind = 2 * num_base + (3 * num_teuk_modes + mode_i) * init_length + i;
          c3_ind = 3 * num_base + (3 * num_teuk_modes + mode_i) * init_length + i;

          L_mode_im_y[i] = interp_array[y_ind];
          L_mode_im_c1[i] = interp_array[c1_ind];
          L_mode_im_c2[i] = interp_array[c2_ind];
          L_mode_im_c3[i] = interp_array[c3_ind];

          y_ind = 0 * num_base + (4 * num_teuk_modes) * init_length + i;
          c1_ind = 1 * num_base + (4 * num_teuk_modes) * init_length + i;
          c2_ind = 2 * num_base + (4 * num_teuk_modes) * init_length + i;
          c3_ind = 3 * num_base + (4 * num_teuk_modes) * init_length + i;

          Phi_phi_y[i] = interp_array[y_ind];
          Phi_phi_c1[i] = interp_array[c1_ind];
          Phi_phi_c2[i] = interp_array[c2_ind];
          Phi_phi_c3[i] = interp_array[c3_ind];

          y_ind = 0 * num_base + (1 + 4 * num_teuk_modes) * init_length + i;
          c1_ind = 1 * num_base + (1 + 4 * num_teuk_modes) * init_length + i;
          c2_ind = 2 * num_base + (1 + 4 * num_teuk_modes) * init_length + i;
          c3_ind = 3 * num_base + (1 + 4 * num_teuk_modes) * init_length + i;

          Phi_theta_y[i] = interp_array[y_ind];
          Phi_theta_c1[i] = interp_array[c1_ind];
          Phi_theta_c2[i] = interp_array[c2_ind];
          Phi_theta_c3[i] = interp_array[c3_ind];

          y_ind = 0 * num_base + (2 + 4 * num_teuk_modes) * init_length + i;
          c1_ind = 1 * num_base + (2 + 4 * num_teuk_modes) * init_length + i;
          c2_ind = 2 * num_base + (2 + 4 * num_teuk_modes) * init_length + i;
          c3_ind = 3 * num_base + (2 + 4 * num_teuk_modes) * init_length + i;

          Phi_r_y[i] = interp_array[y_ind];
          Phi_r_c1[i] = interp_array[c1_ind];
          Phi_r_c2[i] = interp_array[c2_ind];
          Phi_r_c3[i] = interp_array[c3_ind];
      }

      CUDA_SYNC_THREADS;

      #ifdef __HIPCC__

      start = threadIdx.x + blockDim.x * blockIdx.x;
      diff = blockDim.x * gridDim.x;

      #else

      start = 0;
      diff = 1;
      #ifdef __USE_OMP__
      #pragma omp parallel for
      #endif // __USE_OMP__
      #endif
      for (int i = start; i < data_length; i += diff)
      {
          int ind_i = interval_inds[i];
          double start_t = old_time[ind_i];
          
          double R_mode_re_y_i = R_mode_re_y[ind_i];
          double R_mode_re_c1_i = R_mode_re_c1[ind_i];
          double R_mode_re_c2_i = R_mode_re_c2[ind_i];
          double R_mode_re_c3_i = R_mode_re_c3[ind_i];

          double R_mode_im_y_i = R_mode_im_y[ind_i];
          double R_mode_im_c1_i = R_mode_im_c1[ind_i];
          double R_mode_im_c2_i = R_mode_im_c2[ind_i];
          double R_mode_im_c3_i = R_mode_im_c3[ind_i];

          double L_mode_re_y_i = L_mode_re_y[ind_i];
          double L_mode_re_c1_i = L_mode_re_c1[ind_i];
          double L_mode_re_c2_i = L_mode_re_c2[ind_i];
          double L_mode_re_c3_i = L_mode_re_c3[ind_i];

          double L_mode_im_y_i = L_mode_im_y[ind_i];
          double L_mode_im_c1_i = L_mode_im_c1[ind_i];
          double L_mode_im_c2_i = L_mode_im_c2[ind_i];
          double L_mode_im_c3_i = L_mode_im_c3[ind_i];

          double pp_y = Phi_phi_y[ind_i];
          double pp_c1 = Phi_phi_c1[ind_i];
          double pp_c2 = Phi_phi_c2[ind_i];
          double pp_c3 = Phi_phi_c3[ind_i];

          double pt_y = Phi_theta_y[ind_i];
          double pt_c1 = Phi_theta_c1[ind_i];
          double pt_c2 = Phi_theta_c2[ind_i];
          double pt_c3 = Phi_theta_c3[ind_i];

          double pr_y = Phi_r_y[ind_i];
          double pr_c1 = Phi_r_c1[ind_i];
          double pr_c2 = Phi_r_c2[ind_i];
          double pr_c3 = Phi_r_c3[ind_i];
          // determine interpolation information
          double t = delta_t*i;
          double x = t - start_t;
          double x2 = x*x;
          double x3 = x*x2;

            // get mode values at this timestep
            double R_mode_re = R_mode_re_y_i + R_mode_re_c1_i * x + R_mode_re_c2_i * x2  + R_mode_re_c3_i * x3;
            double R_mode_im = R_mode_im_y_i + R_mode_im_c1_i * x + R_mode_im_c2_i * x2  + R_mode_im_c3_i * x3;
            double L_mode_re = L_mode_re_y_i + L_mode_re_c1_i * x + L_mode_re_c2_i * x2  + L_mode_re_c3_i * x3;
            double L_mode_im = L_mode_im_y_i + L_mode_im_c1_i * x + L_mode_im_c2_i * x2  + L_mode_im_c3_i * x3;

            // get phases at this timestep
            double Phi_phi_i = pp_y + pp_c1 * x + pp_c2 * x2  + pp_c3 * x3;
            double Phi_theta_i = pt_y + pt_c1 * x + pt_c2 * x2 + pt_c3 * x3;
            double Phi_r_i = pr_y + pr_c1 * x + pr_c2 * x2  + pr_c3 * x3;

            cmplx R_amp(R_mode_re, R_mode_im);
            cmplx L_amp(L_mode_re, L_mode_im);

            cmplx R_tmp = get_mode_value_generic(R_amp, Phi_phi_i, Phi_r_i, Phi_theta_i, m, k, n);

            
            cmplx L_tmp(0.0, 0.0);
            if (m + k + n != 0)
            {
              L_tmp = get_mode_value_generic(L_amp, Phi_phi_i, Phi_r_i, Phi_theta_i, -m, -k, -n);
            }

            cmplx wave_mode_out(0.0, 0.0);
            if (!separate_modes)
            {
              wave_mode_out = R_tmp + L_tmp;

              // fill waveform
              #ifdef __HIPCC__
              atomicAddComplex(&waveform[i], wave_mode_out);
              #else
              waveform[i] += wave_mode_out;
              #endif
            }
            else 
            {
              waveform[mode_i * data_length + i] = R_tmp;
              waveform[(num_teuk_modes * data_length) + mode_i * data_length + i] = L_tmp;
            }
      }
    }          
    CUDA_SYNC_THREADS;
}


#include "Utility.hh"

// function for building interpolated EMRI waveform from python
void get_waveform_generic(cmplx *waveform,
             double *interp_array,
              int *m_arr_in, int *k_arr_in, int *n_arr_in, int num_teuk_modes,
              double delta_t, double *old_time_arr, int init_length, int data_length, int *interval_inds, bool separate_modes)
{

     int NUM_THREADS = 256;

     if (init_length > MAX_SPLINE_POINTS)
     {
        char str[1000];
        sprintf(str, "Number of initial points is more than allowed for interpolated summation. (%d > %d)", init_length, MAX_SPLINE_POINTS);
        throw std::invalid_argument(str);
     }

     #ifdef __HIPCC__

      int num_blocks = std::ceil((data_length + NUM_THREADS -1)/NUM_THREADS);

      dim3 gridDim(num_blocks, num_teuk_modes);
      // launch one worker kernel per stream
      make_generic_kerr_waveform<<<gridDim, NUM_THREADS>>>(waveform,
             interp_array,
              m_arr_in, k_arr_in, n_arr_in, num_teuk_modes,
              delta_t, old_time_arr, init_length, data_length, interval_inds, separate_modes);
      hipDeviceSynchronize();
      gpuErrchk(hipGetLastError());
      
      #else

         // CPU waveform generation
         make_generic_kerr_waveform(waveform,
             interp_array,
              m_arr_in, k_arr_in, n_arr_in, num_teuk_modes,
              delta_t, old_time_arr, init_length, data_length, interval_inds, separate_modes);
         
        #endif

}



// build mode value with specific phase and amplitude values; mode indexes; and spherical harmonics
CUDA_CALLABLE_MEMBER
cmplx get_mode_value_generic_tf(cmplx teuk_mode, fod Phi_phi, fod Phi_theta, fod Phi_r, int m, int k, int n){
    cmplx minus_I(0.0, -1.0);
    fod phase = m * Phi_phi + k * Phi_theta + n * Phi_r;
    cmplx out = teuk_mode * gcmplx::exp(minus_I*phase);
    return out;
}

CUDA_CALLABLE_MEMBER
cmplx DirichletKernel(double f, double T, double dt)
{
    cmplx I(0.0, 1.0);
    double num = sin(M_PI * f * T);
    double denom = sin(M_PI * f * dt);
    double out;
    if (denom == 0.0)
    {
      out = 1.0;
    }
    else
    {
      out = num / denom;
    }
    return gcmplx::exp(-I * M_PI * f * (T - dt)) * out;
}
    
CUDA_CALLABLE_MEMBER
cmplx get_DFT(double A, int n, double dt, double f, double f0, double phi0)
{
  cmplx I(0.0, 1.0);
  double T = n * dt;
  return (
        A
        * (
            DirichletKernel(f - f0, T, dt) * gcmplx::exp(-I * phi0)
            + DirichletKernel(f + f0, T, dt) * gcmplx::exp(+I * phi0)
        )
        / 2.
    );
}
    

// make a waveform in parallel
// this uses an efficient summation by loading mode information into shared memory
// shared memory is leveraged heavily
#define MAX_SPLINE_POINTS 210
CUDA_KERNEL
void make_generic_kerr_waveform_tf(cmplx *waveform,
             double *interp_array,
              int *m_arr_in, int *k_arr_in, int *n_arr_in, int num_teuk_modes,
              double delta_t, double start_t, double *old_time_arr, int init_length, int data_length, int *interval_inds, bool separate_modes, int num_windows, int num_per_window, int inds_left_right, int freq_length, bool include_L){

    int num_pars = 6;

    #ifdef __HIPCC__
    extern __shared__  unsigned char shared_mem[];
    cmplx* window_output = (cmplx *) shared_mem;
    #else
    cmplx window_output_temp[freq_length];
    cmplx* window_output = &window_output_temp[0];
    #endif
  

    cmplx complexI(0.0, 1.0);
    double re_y, re_c1, re_c2, re_c3, im_y, im_c1, im_c2, im_c3;
     CUDA_SHARED double pp_y, pp_c1, pp_c2, pp_c3, pr_y, pr_c1, pr_c2, pr_c3;

     // declare all the shared memory
     // MAX_MODES_BLOCK is fixed based on shared memory
     double old_time;

     double R_mode_re_y;
     double R_mode_re_c1;
     double R_mode_re_c2;
     double R_mode_re_c3;

     double R_mode_im_y;
     double R_mode_im_c1;
     double R_mode_im_c2;
     double R_mode_im_c3;

     double L_mode_re_y;
     double L_mode_re_c1;
     double L_mode_re_c2;
     double L_mode_re_c3;

     double L_mode_im_y;
     double L_mode_im_c1;
     double L_mode_im_c2;
     double L_mode_im_c3;

     double Phi_phi_y;
     double Phi_phi_c1;
     double Phi_phi_c2;
     double Phi_phi_c3;

     double Phi_theta_y;
     double Phi_theta_c1;
     double Phi_theta_c2;
     double Phi_theta_c3;

     double Phi_r_y;
     double Phi_r_c1;
     double Phi_r_c2;
     double Phi_r_c3;
       
     // number of splines
     int num_base = (4 * num_teuk_modes + num_pars) * init_length;
     int total_middle = (4 * num_teuk_modes + num_pars);

    double T_window = num_per_window * delta_t;
    double df_window = 1. / T_window;
    double Phi_phi, Phi_theta, Phi_r, f_phi_y, f_phi_c1, f_phi_c2, f_phi_c3, f_theta_y, f_theta_c1, f_theta_c2, f_theta_c3, f_r_y, f_r_c1, f_r_c2, f_r_c3, R_mode_re, R_mode_im, L_mode_re, L_mode_im, f_phi, f_theta, f_r;

    int start_ind;

    #ifdef __HIPCC__

    int start = blockIdx.x;
    int diff = gridDim.x;

    #else

    int start = 0;
    int diff = 1;
    #ifdef __USE_OMP__
    #pragma omp parallel for
    #endif // __USE_OMP__
    #endif
    for (int t_i = start; t_i < init_length; t_i += diff) 
    {
      
      double t_new = start_t + t_i * delta_t;
      int ind_here = interval_inds[t_i];
      double t_old = old_time_arr[ind_here];
      double x = t_new - t_old;
      double x2 = x * x;
      double x3 = x2 * x;

      int y_ind = 0 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 0);
      int c1_ind = 1 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 0);
      int c2_ind = 2 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 0);
      int c3_ind = 3 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 0);

      Phi_phi_y = interp_array[y_ind];
      Phi_phi_c1 = interp_array[c1_ind];
      Phi_phi_c2 = interp_array[c2_ind];
      Phi_phi_c3 = interp_array[c3_ind];

      Phi_phi = Phi_phi_y + Phi_phi_c1 * x + Phi_phi_c2 * x2 + Phi_phi_c3 * x3;

      y_ind = 0 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 1);
      c1_ind = 1 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 1);
      c2_ind = 2 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 1);
      c3_ind = 3 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 1);

      Phi_theta_y = interp_array[y_ind];
      Phi_theta_c1 = interp_array[c1_ind];
      Phi_theta_c2 = interp_array[c2_ind];
      Phi_theta_c3 = interp_array[c3_ind];

      Phi_theta = Phi_theta_y + Phi_theta_c1 * x + Phi_theta_c2 * x2 + Phi_theta_c3 * x3;

      y_ind = 0 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 2);
      c1_ind = 1 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 2);
      c2_ind = 2 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 2);
      c3_ind = 3 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 2);

      Phi_r_y = interp_array[y_ind];
      Phi_r_c1 = interp_array[c1_ind];
      Phi_r_c2 = interp_array[c2_ind];
      Phi_r_c3 = interp_array[c3_ind];

      Phi_r = Phi_r_y + Phi_r_c1 * x + Phi_r_c2 * x2 + Phi_r_c3 * x3;

      y_ind = 0 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 3);
      c1_ind = 1 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 3);
      c2_ind = 2 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 3);
      c3_ind = 3 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 3);

      f_phi_y = interp_array[y_ind];
      f_phi_c1 = interp_array[c1_ind];
      f_phi_c2 = interp_array[c2_ind];
      f_phi_c3 = interp_array[c3_ind];

      f_phi = f_phi_y + f_phi_c1 * x + f_phi_c2 * x2 + f_phi_c3 * x3;

      y_ind = 0 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 4);
      c1_ind = 1 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 4);
      c2_ind = 2 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 4);
      c3_ind = 3 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 4);

      f_theta_y = interp_array[y_ind];
      f_theta_c1 = interp_array[c1_ind];
      f_theta_c2 = interp_array[c2_ind];
      f_theta_c3 = interp_array[c3_ind];

      f_theta = f_theta_y + f_theta_c1 * x + f_theta_c2 * x2 + f_theta_c3 * x3;

      y_ind = 0 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 5);
      c1_ind = 1 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 5);
      c2_ind = 2 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 5);
      c3_ind = 3 * num_base + (ind_here * total_middle) + (4 * num_teuk_modes + 5);

      f_r_y = interp_array[y_ind];
      f_r_c1 = interp_array[c1_ind];
      f_r_c2 = interp_array[c2_ind];
      f_r_c3 = interp_array[c3_ind];

      f_r = f_r_y + f_r_c1 * x + f_r_c2 * x2 + f_r_c3 * x3;

      #ifdef __HIPCC__

      int start2 = threadIdx.x;
      int diff2 = blockDim.x;

      #else

      int start2 = 0;
      int diff2 = 1;
      #ifdef __USE_OMP__
      #pragma omp parallel for
      #endif // __USE_OMP__
      #endif
      for (int mode_i = start2; mode_i < num_teuk_modes; mode_i += diff2)
      {
          
          int m = m_arr_in[mode_i];
          int k = k_arr_in[mode_i];
          int n = n_arr_in[mode_i];

          y_ind = 0 * num_base + (ind_here * total_middle) + mode_i;
          c1_ind = 1 * num_base + (ind_here * total_middle) + mode_i;
          c2_ind = 2 * num_base + (ind_here * total_middle) + mode_i;
          c3_ind = 3 * num_base + (ind_here * total_middle) + mode_i;

          R_mode_re_y = interp_array[y_ind];
          R_mode_re_c1 = interp_array[c1_ind];
          R_mode_re_c2 = interp_array[c2_ind];
          R_mode_re_c3 = interp_array[c3_ind];

          R_mode_re = R_mode_re_y + R_mode_re_c1 * x + R_mode_re_c2 * x2 + R_mode_re_c3 * x3;

          y_ind = 0 * num_base + (ind_here * total_middle) + num_teuk_modes + mode_i;
          c1_ind = 1 * num_base + (ind_here * total_middle) + num_teuk_modes+ mode_i;
          c2_ind = 2 * num_base + (ind_here * total_middle) + num_teuk_modes+ mode_i;
          c3_ind = 3 * num_base + (ind_here * total_middle) + num_teuk_modes+ mode_i;

          R_mode_im_y = interp_array[y_ind];
          R_mode_im_c1 = interp_array[c1_ind];
          R_mode_im_c2 = interp_array[c2_ind];
          R_mode_im_c3 = interp_array[c3_ind];

          R_mode_im = R_mode_im_y + R_mode_im_c1 * x + R_mode_im_c2 * x2 + R_mode_im_c3 * x3;

          y_ind = 0 * num_base + (ind_here * total_middle) + 2 * num_teuk_modes + mode_i;
          c1_ind = 1 * num_base + (ind_here * total_middle) + 2 * num_teuk_modes + mode_i;
          c2_ind = 2 * num_base + (ind_here * total_middle) + 2 * num_teuk_modes + mode_i;
          c3_ind = 3 * num_base + (ind_here * total_middle) + 2 * num_teuk_modes + mode_i;

          L_mode_re_y = interp_array[y_ind];
          L_mode_re_c1 = interp_array[c1_ind];
          L_mode_re_c2 = interp_array[c2_ind];
          L_mode_re_c3 = interp_array[c3_ind];

          L_mode_re = L_mode_re_y + L_mode_re_c1 * x + L_mode_re_c2 * x2 + L_mode_re_c3 * x3;

          y_ind = 0 * num_base + (ind_here * total_middle) + 3 * num_teuk_modes + mode_i;
          c1_ind = 1 * num_base + (ind_here * total_middle) + 3 * num_teuk_modes + mode_i;
          c2_ind = 2 * num_base + (ind_here * total_middle) + 3 * num_teuk_modes + mode_i;
          c3_ind = 3 * num_base + (ind_here * total_middle) + 3 * num_teuk_modes + mode_i;

          L_mode_im_y = interp_array[y_ind];
          L_mode_im_c1 = interp_array[c1_ind];
          L_mode_im_c2 = interp_array[c2_ind];
          L_mode_im_c3 = interp_array[c3_ind];

          L_mode_im = L_mode_im_y + L_mode_im_c1 * x + L_mode_im_c2 * x2 + L_mode_im_c3 * x3;

          cmplx R_amp(R_mode_re, R_mode_im);
          cmplx L_amp(L_mode_re, L_mode_im);

          double f_mode = m * f_phi + k * f_theta + n * f_r;
          double phase_mode = m * Phi_phi + k * Phi_theta + n * Phi_r;
          
          int closest_f_ind = (int)rint(f_mode / df_window);

          int start_f_ind = closest_f_ind - inds_left_right;
          int end_f_ind = closest_f_ind + inds_left_right;

          if (start_f_ind < 0) start_f_ind = 0;
          if (end_f_ind >= freq_length) end_f_ind = freq_length - 1;

          double f_tmp;
          // must be <=
          for (int j = start_ind; j <= end_f_ind; j += 1)
          {
              f_tmp = j * df_window;
              cmplx sin_term_R = get_DFT(
                    1.0, num_per_window, delta_t, f_tmp, f_mode, phase_mode
              );

              cmplx cos_term_R = get_DFT(
                    1.0, num_per_window, delta_t, f_tmp, f_mode, phase_mode - M_PI / 4.
              );

              cmplx R_tmp_plus = (R_amp.real() * cos_term_R + R_amp.imag() * sin_term_R);
              cmplx R_tmp_cross = (R_amp.real() * sin_term_R - R_amp.imag() * cos_term_R);

              cmplx L_tmp_plus, L_tmp_cross, sin_term_L, cos_term_L;
              if (include_L)
              {
                  sin_term_L = get_DFT(
                    1.0, num_per_window, delta_t, f_tmp, -f_mode, -phase_mode
                );
                  cos_term_L = get_DFT(
                    1.0, num_per_window, delta_t, f_tmp, -f_mode, -phase_mode - M_PI / 4.
                );
                L_tmp_plus = (L_amp.real() * cos_term_L + L_amp.imag() * sin_term_L);
                L_tmp_cross = (L_amp.real() * sin_term_L - L_amp.imag() * cos_term_L);
              }
                
              else
              {
                L_tmp_plus = cmplx(0.0, 0.0);
                L_tmp_cross = cmplx(0.0, 0.0);
              }
              window_output[j] = R_amp_plus + L_amp_plus;
              window_output[f_length + j] = R_amp_cross + L_amp_cross;

          }
          
      }
      CUDA_SYNC_THREADS;
      for (int j = start2; j < 2 * freq_length; j += diff2)
      {
          // does both plus and cross
          waveform[t_i * (2 * freq_length) + j] = window_output[j];
      }
    }
}


#include "Utility.hh"

// function for building interpolated EMRI waveform from python
void get_waveform_tf_generic(cmplx *waveform,
             double *interp_array,
              int *m_arr_in, int *k_arr_in, int *n_arr_in, int num_teuk_modes,
              double delta_t, double start_t, double *old_time_arr, int init_length, int data_length, int *interval_inds, bool separate_modes, int num_windows, int num_per_window, int inds_left_right, int freq_length, bool include_L)
{

     int NUM_THREADS = 256;

     #ifdef __HIPCC__

     auto shared_memory_size = freq_length * sizeof(cmplx);

    // Increase max shared memory if needed
    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(
        make_generic_kerr_waveform_tf),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        shared_memory_size));

      dim3 gridDim(num_windows);
      // launch one worker kernel per stream
      make_generic_kerr_waveform_tf<<<gridDim, NUM_THREADS, shared_memory_size>>>(waveform,
             interp_array,
              m_arr_in, k_arr_in, n_arr_in, num_teuk_modes,
              delta_t, start_t, old_time_arr, init_length, data_length, interval_inds, separate_modes, num_windows, num_per_window, inds_left_right, freq_length, include_L);
      hipDeviceSynchronize();
      gpuErrchk(hipGetLastError());
      
      #else

         // CPU waveform generation
         make_generic_kerr_waveform_tf(waveform,
             interp_array,
              m_arr_in, k_arr_in, n_arr_in, num_teuk_modes,
              delta_t, start_t, old_time_arr, init_length, data_length, interval_inds, separate_modes, num_windows, num_per_window, inds_left_right, freq_length, include_L);
         
        #endif

}

